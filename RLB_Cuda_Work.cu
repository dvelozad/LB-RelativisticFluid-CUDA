#include<iostream>
#include<fstream>
#include<stdio.h>
#include<hip/hip_runtime.h>


using namespace std;

#define M 20
#define N 10
#define W 10

#define Q 19

#define BLOCKSIZE_x 5
#define BLOCKSIZE_y 5
#define BLOCKSIZE_z 5


const int Mx=(M + BLOCKSIZE_x -1)/BLOCKSIZE_x;
const int My=(N + BLOCKSIZE_y -1)/BLOCKSIZE_y;
const int Mz=(W + BLOCKSIZE_z -1)/BLOCKSIZE_z;

const int cl = 1;
const int C = 1;
const float tau = 7.5;

__constant__ float d_w[Q];
__constant__ int d_Vx[Q];
__constant__ int d_Vy[Q];
__constant__ int d_Vz[Q];

__constant__ int d_cl = cl;
__constant__ int d_C = C;
__constant__ float d_tau = tau;

__constant__ int d_cm_x = 100;
__constant__ int d_cm_y = 50;
__constant__ int d_cm_z = 50;

__constant__ int d_R = 10;


/******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

//-----------------------------------------------------------------------
//----------------------------Macroscopic Constrains---------------------
//-----------------------------------------------------------------------

//____________________________________Pressure__________________________________
__device__ float d_P(float g0,float g1,float g2,float g3,float g4,
                     float g5,float g6,float g7,float g8,float g9,
                     float g10,float g11,float g12,float g13,float g14,
                     float g15,float g16,float g17,float g18){
  int i,j; float sum1=0, sum2=0;
  float g_aux[19] = {g0,g1,g2,g3,g4,g5,g6,g7,g8,g9,g10,g11,g12,g13,g14,g15,g16,g17,g18};
  for(i=0;i<Q;i++){
    sum1 += g_aux[i];
    for(j=0;j<Q;j++){
      sum2 += (g_aux[i]*g_aux[j]*(d_Vx[i]*d_Vx[j]+d_Vy[i]*d_Vy[j]+d_Vz[i]*d_Vz[j]));
    }
  }
  return -(1./3.)*sum1 + (1./3.)*sqrt(-3.*sum2 + 4.*sum1*sum1);
}
//_________________________________Energy Density______________________________
__device__ float d_rho(float g0,float g1,float g2,float g3,float g4,
                       float g5,float g6,float g7,float g8,float g9,
                       float g10,float g11,float g12,float g13,float g14,
                       float g15,float g16,float g17,float g18){
  return 3.*d_P(g0,g1,g2,g3,g4,g5,g6,g7,g8,g9,g10,g11,g12,g13,g14,g15,g16,g17,g18);
}
//__________________________________Velocity Field______________________________
__device__ float d_Ux(float g0,float g1,float g2,float g3,float g4,
                      float g5,float g6,float g7,float g8,float g9,
                      float g10,float g11,float g12,float g13,float g14,
                      float g15,float g16,float g17,float g18){
  
  float sum1=0, sum2=0;
  
  sum2 = g0*d_Vx[0]+g1*d_Vx[1]+g2*d_Vx[2]+g3*d_Vx[3]+g4*d_Vx[4]+g5*d_Vx[5]+g6*d_Vx[6]+g7*d_Vx[7]+g8*d_Vx[8]+g9*d_Vx[9]+g10*d_Vx[10]+g11*d_Vx[11]+g12*d_Vx[12]+g13*d_Vx[13]+g14*d_Vx[14]+g15*d_Vx[15]+g16*d_Vx[16]+g17*d_Vx[17]+g18*d_Vx[18];

  sum1 = g0+g1+g2+g3+g4+g5+g6+g7+g8+g9+g10+g11+g12+g13+g14+g15+g16+g17+g18;
  
  return 3.*sum2/(3.*sum1 + 3.*d_P(g0,g1,g2,g3,g4,g5,g6,g7,g8,g9,g10,g11,g12,g13,g14,g15,g16,g17,g18));
}
//--------------//
__device__ float d_Uy(float g0,float g1,float g2,float g3,float g4,
                      float g5,float g6,float g7,float g8,float g9,
                      float g10,float g11,float g12,float g13,float g14,
                      float g15,float g16,float g17,float g18){
  float sum1=0, sum2=0;

  sum2 = g0*d_Vy[0]+g1*d_Vy[1]+g2*d_Vy[2]+g3*d_Vy[3]+g4*d_Vy[4]+g5*d_Vy[5]+g6*d_Vy[6]+g7*d_Vy[7]+g8*d_Vy[8]+g9*d_Vy[9]+g10*d_Vy[10]+g11*d_Vy[11]+g12*d_Vy[12]+g13*d_Vy[13]+g14*d_Vy[14]+g15*d_Vy[15]+g16*d_Vy[16]+g17*d_Vy[17]+g18*d_Vy[18];

  sum1 = g0+g1+g2+g3+g4+g5+g6+g7+g8+g9+g10+g11+g12+g13+g14+g15+g16+g17+g18;
  
  return 3.*sum2/(3.*sum1 + 3.*d_P(g0,g1,g2,g3,g4,g5,g6,g7,g8,g9,g10,g11,g12,g13,g14,g15,g16,g17,g18));
}
//-------------//
__device__ float d_Uz(float g0,float g1,float g2,float g3,float g4,
                      float g5,float g6,float g7,float g8,float g9,
                      float g10,float g11,float g12,float g13,float g14,
                      float g15,float g16,float g17,float g18){
  float sum1=0, sum2=0;

  sum2 = g0*d_Vz[0]+g1*d_Vz[1]+g2*d_Vz[2]+g3*d_Vz[3]+g4*d_Vz[4]+g5*d_Vz[5]+g6*d_Vz[6]+g7*d_Vz[7]+g8*d_Vz[8]+g9*d_Vz[9]+g10*d_Vz[10]+g11*d_Vz[11]+g12*d_Vz[12]+g13*d_Vz[13]+g14*d_Vz[14]+g15*d_Vz[15]+g16*d_Vz[16]+g17*d_Vz[17]+g18*d_Vz[18];

  sum1 = g0+g1+g2+g3+g4+g5+g6+g7+g8+g9+g10+g11+g12+g13+g14+g15+g16+g17+g18;
  
  return 3.*sum2/(3.*sum1 + 3.*d_P(g0,g1,g2,g3,g4,g5,g6,g7,g8,g9,g10,g11,g12,g13,g14,g15,g16,g17,g18));
}
//___________________________________Gamma___________________________________
__device__ float d_gamma(float Ux0,float Uy0,float Uz0){
  float U2;
  U2 = Ux0*Ux0 + Uy0*Uy0 + Uz0*Uz0;
  return 1./sqrt(1.-(U2/(d_C*d_C)));
}
//______________________________Particle density______________________________
__device__ float d_n(float f0,float f1,float f2,float f3,float f4,
                     float f5,float f6,float f7,float f8,float f9,
                     float f10,float f11,float f12,float f13,float f14,
                     float f15,float f16,float f17,float f18,
                     float Ux0,float Uy0,float Uz0){
  float sum = 0;
  sum = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+f10+f11+f12+f13+f14+f15+f16+f17+f18;
  return sum/d_gamma(Ux0,Uy0,Uz0);
}
//------------------------------------------------------------------------------
//--------------------------------Equilibrium Functions-------------------------
//------------------------------------------------------------------------------

__device__ float d_feq(int i,float n0,float Ux0,float Uy0,float Uz0){
  float y,U2,UdotV;

  y = d_gamma(Ux0,Uy0,Uz0);
  UdotV = Ux0*d_Vx[i]+Uy0*d_Vy[i]+Uz0*d_Vz[i];
  U2 = Ux0*Ux0 + Uy0*Uy0 + Uz0*Uz0;

  return d_w[i]*n0*y*(1.+3.*UdotV/(d_cl*d_cl) + (9./2.)*(UdotV*UdotV)/(d_cl*d_cl*d_cl*d_cl) - (3./2.)*(U2/(d_cl*d_cl)));
}
__device__ float d_geq(int i,float rho0,float P0,float Ux0,float Uy0,float Uz0){
  float y2,UdotV,U2;

  y2 = d_gamma(Ux0,Uy0,Uz0)*d_gamma(Ux0,Uy0,Uz0);
  UdotV = Ux0*d_Vx[i]+Uy0*d_Vy[i]+Uz0*d_Vz[i];
  U2 = Ux0*Ux0 + Uy0*Uy0 + Uz0*Uz0;
  
  if(i == 0){
    return 3.*P0*d_w[0]*y2*(4. - (2.+ d_cl*d_cl)/(y2*d_cl*d_cl) - 2.*(U2/(d_cl*d_cl)));
  }else{
    return 3.*d_w[i]*P0*y2*( 1./(y2*d_cl*d_cl) + 4.*UdotV/(d_cl*d_cl) + 6.*(UdotV*UdotV)/(d_cl*d_cl*d_cl*d_cl) - 2.*(U2/(d_cl*d_cl)) );
  }
}
/**********************************************/
__global__ void op_indv_advection(hipPitchedPtr devPitchedPtrI,hipPitchedPtr devPitchedPtrInew,int I)
{  
  //printf("\o/");
  int ix =  blockIdx.x*blockDim.x+threadIdx.x;
  int iy =  blockIdx.y*blockDim.y+threadIdx.y;
  int iz =  blockIdx.z*blockDim.z+threadIdx.z;

  //--------------------------------------------
  int _ix = (M + ix + d_Vx[I])%M;
  int _iy = (N + iy + d_Vy[I])%N;
  int _iz = (W + iz + d_Vz[I])%W;
  
  char* devPtrI = (char*) devPitchedPtrI.ptr;
  size_t pitchI = devPitchedPtrI.pitch;
  size_t slicePitchI = pitchI * N;
  
  char* sliceI = devPtrI + _iz * slicePitchI;
  float* fI = (float*)(sliceI + _iy * pitchI);
  
  char* devPtrInew = (char*) devPitchedPtrInew.ptr;
  size_t pitchInew = devPitchedPtrInew.pitch;
  size_t slicePitchInew = pitchInew * N;
  
  char* sliceInew = devPtrInew + iz * slicePitchInew;
  float* fInew = (float*)(sliceInew + iy * pitchInew);
  
  //printf("| %i ",_iz);
  //if((_ix != M & _ix != -1) & (_iy != N & _iy != -1) & (_iz != W & _iz != -1)){
  if((ix >= 1 & ix < M-1) & (iy >= 1 & iy < N-1)  & (iz >= 1 & iz < W-1) ){
    fI[_ix] = fInew[ix];
  }
  //}
}
__global__ void d_collition(hipPitchedPtr devpitchf0,hipPitchedPtr devpitchf0new,hipPitchedPtr devpitchg0,hipPitchedPtr devpitchg0new,
                            hipPitchedPtr devpitchf1,hipPitchedPtr devpitchf1new,hipPitchedPtr devpitchg1,hipPitchedPtr devpitchg1new,
                            hipPitchedPtr devpitchf2,hipPitchedPtr devpitchf2new,hipPitchedPtr devpitchg2,hipPitchedPtr devpitchg2new,
                            hipPitchedPtr devpitchf3,hipPitchedPtr devpitchf3new,hipPitchedPtr devpitchg3,hipPitchedPtr devpitchg3new,
                            hipPitchedPtr devpitchf4,hipPitchedPtr devpitchf4new,hipPitchedPtr devpitchg4,hipPitchedPtr devpitchg4new,
                            hipPitchedPtr devpitchf5,hipPitchedPtr devpitchf5new,hipPitchedPtr devpitchg5,hipPitchedPtr devpitchg5new,
                            hipPitchedPtr devpitchf6,hipPitchedPtr devpitchf6new,hipPitchedPtr devpitchg6,hipPitchedPtr devpitchg6new,
                            hipPitchedPtr devpitchf7,hipPitchedPtr devpitchf7new,hipPitchedPtr devpitchg7,hipPitchedPtr devpitchg7new,
                            hipPitchedPtr devpitchf8,hipPitchedPtr devpitchf8new,hipPitchedPtr devpitchg8,hipPitchedPtr devpitchg8new,
                            hipPitchedPtr devpitchf9,hipPitchedPtr devpitchf9new,hipPitchedPtr devpitchg9,hipPitchedPtr devpitchg9new,
                            hipPitchedPtr devpitchf10,hipPitchedPtr devpitchf10new,hipPitchedPtr devpitchg10,hipPitchedPtr devpitchg10new,
                            hipPitchedPtr devpitchf11,hipPitchedPtr devpitchf11new,hipPitchedPtr devpitchg11,hipPitchedPtr devpitchg11new,
                            hipPitchedPtr devpitchf12,hipPitchedPtr devpitchf12new,hipPitchedPtr devpitchg12,hipPitchedPtr devpitchg12new,
                            hipPitchedPtr devpitchf13,hipPitchedPtr devpitchf13new,hipPitchedPtr devpitchg13,hipPitchedPtr devpitchg13new,
                            hipPitchedPtr devpitchf14,hipPitchedPtr devpitchf14new,hipPitchedPtr devpitchg14,hipPitchedPtr devpitchg14new,
                            hipPitchedPtr devpitchf15,hipPitchedPtr devpitchf15new,hipPitchedPtr devpitchg15,hipPitchedPtr devpitchg15new,
                            hipPitchedPtr devpitchf16,hipPitchedPtr devpitchf16new,hipPitchedPtr devpitchg16,hipPitchedPtr devpitchg16new,
                            hipPitchedPtr devpitchf17,hipPitchedPtr devpitchf17new,hipPitchedPtr devpitchg17,hipPitchedPtr devpitchg17new,
                            hipPitchedPtr devpitchf18,hipPitchedPtr devpitchf18new,hipPitchedPtr devpitchg18,hipPitchedPtr devpitchg18new){
  
  int ix =  blockIdx.x*blockDim.x+threadIdx.x;
  int iy =  blockIdx.y*blockDim.y+threadIdx.y;
  int iz =  blockIdx.z*blockDim.z+threadIdx.z;
  //printf("|%i",ix);
  //--------------------------------------------
  //--------------------------------------------
  
  char* devPtrf0 = (char*) devpitchf0.ptr;
  size_t pitchf0 = devpitchf0.pitch;
  size_t slicePitchf0 = pitchf0 * N;
  
  char* slicef0 = devPtrf0 + iz * slicePitchf0;
  float* f0 = (float*)(slicef0 + iy * pitchf0);
  
  char* devPtrf0new = (char*) devpitchf0new.ptr;
  size_t pitchf0new = devpitchf0new.pitch;
  size_t slicePitchf0new = pitchf0new * N;
  
  char* slicef0new = devPtrf0new + iz * slicePitchf0new;
  float* f0new = (float*)(slicef0new + iy * pitchf0new);
  //---------------------------------------------------
  char* devPtrg0 = (char*) devpitchg0.ptr;
  size_t pitchg0 = devpitchg0.pitch;
  size_t slicePitchg0 = pitchg0 * N;
  
  char* sliceg0 = devPtrg0 + iz * slicePitchg0;
  float* g0 = (float*)(sliceg0 + iy * pitchg0);
  
  char* devPtrg0new = (char*) devpitchg0new.ptr;
  size_t pitchg0new = devpitchg0new.pitch;
  size_t slicePitchg0new = pitchg0new * N;
  
  char* sliceg0new = devPtrg0new + iz * slicePitchg0new;
  float* g0new = (float*)(sliceg0new + iy * pitchg0new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf1 = (char*) devpitchf1.ptr;
  size_t pitchf1 = devpitchf1.pitch;
  size_t slicePitchf1 = pitchf1 * N;
  
  char* slicef1 = devPtrf1 + iz * slicePitchf1;
  float* f1 = (float*)(slicef1 + iy * pitchf1);
  
  char* devPtrf1new = (char*) devpitchf1new.ptr;
  size_t pitchf1new = devpitchf1new.pitch;
  size_t slicePitchf1new = pitchf1new * N;
  
  char* slicef1new = devPtrf1new + iz * slicePitchf1new;
  float* f1new = (float*)(slicef1new + iy * pitchf1new);
  //---------------------------------------------------
  char* devPtrg1 = (char*) devpitchg1.ptr;
  size_t pitchg1 = devpitchg1.pitch;
  size_t slicePitchg1 = pitchg1 * N;
  
  char* sliceg1 = devPtrg1 + iz * slicePitchg1;
  float* g1 = (float*)(sliceg1 + iy * pitchg1);
  
  char* devPtrg1new = (char*) devpitchg1new.ptr;
  size_t pitchg1new = devpitchg1new.pitch;
  size_t slicePitchg1new = pitchg1new * N;
  
  char* sliceg1new = devPtrg1new + iz * slicePitchg1new;
  float* g1new = (float*)(sliceg1new + iy * pitchg1new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf2 = (char*) devpitchf2.ptr;
  size_t pitchf2 = devpitchf2.pitch;
  size_t slicePitchf2 = pitchf2 * N;
  
  char* slicef2 = devPtrf2 + iz * slicePitchf2;
  float* f2 = (float*)(slicef2 + iy * pitchf2);
  
  char* devPtrf2new = (char*) devpitchf2new.ptr;
  size_t pitchf2new = devpitchf2new.pitch;
  size_t slicePitchf2new = pitchf2new * N;
  
  char* slicef2new = devPtrf2new + iz * slicePitchf2new;
  float* f2new = (float*)(slicef2new + iy * pitchf2new);
  //---------------------------------------------------
  char* devPtrg2 = (char*) devpitchg2.ptr;
  size_t pitchg2 = devpitchg2.pitch;
  size_t slicePitchg2 = pitchg2 * N;
  
  char* sliceg2 = devPtrg2 + iz * slicePitchg2;
  float* g2 = (float*)(sliceg2 + iy * pitchg2);
  
  char* devPtrg2new = (char*) devpitchg2new.ptr;
  size_t pitchg2new = devpitchg2new.pitch;
  size_t slicePitchg2new = pitchg2new * N;
  
  char* sliceg2new = devPtrg2new + iz * slicePitchg2new;
  float* g2new = (float*)(sliceg2new + iy * pitchg2new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf3 = (char*) devpitchf3.ptr;
  size_t pitchf3 = devpitchf3.pitch;
  size_t slicePitchf3 = pitchf3 * N;
  
  char* slicef3 = devPtrf3 + iz * slicePitchf3;
  float* f3 = (float*)(slicef3 + iy * pitchf3);
  
  char* devPtrf3new = (char*) devpitchf3new.ptr;
  size_t pitchf3new = devpitchf3new.pitch;
  size_t slicePitchf3new = pitchf3new * N;
  
  char* slicef3new = devPtrf3new + iz * slicePitchf3new;
  float* f3new = (float*)(slicef3new + iy * pitchf3new);
  //---------------------------------------------------
  char* devPtrg3 = (char*) devpitchg3.ptr;
  size_t pitchg3 = devpitchg3.pitch;
  size_t slicePitchg3 = pitchg3 * N;
  
  char* sliceg3 = devPtrg3 + iz * slicePitchg3;
  float* g3 = (float*)(sliceg3 + iy * pitchg3);
  
  char* devPtrg3new = (char*) devpitchg3new.ptr;
  size_t pitchg3new = devpitchg3new.pitch;
  size_t slicePitchg3new = pitchg3new * N;
  
  char* sliceg3new = devPtrg3new + iz * slicePitchg3new;
  float* g3new = (float*)(sliceg3new + iy * pitchg3new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf4 = (char*) devpitchf4.ptr;
  size_t pitchf4 = devpitchf4.pitch;
  size_t slicePitchf4 = pitchf4 * N;
  
  char* slicef4 = devPtrf4 + iz * slicePitchf4;
  float* f4 = (float*)(slicef4 + iy * pitchf4);
  
  char* devPtrf4new = (char*) devpitchf4new.ptr;
  size_t pitchf4new = devpitchf4new.pitch;
  size_t slicePitchf4new = pitchf4new * N;
  
  char* slicef4new = devPtrf4new + iz * slicePitchf4new;
  float* f4new = (float*)(slicef4new + iy * pitchf4new);
  //---------------------------------------------------
  char* devPtrg4 = (char*) devpitchg4.ptr;
  size_t pitchg4 = devpitchg4.pitch;
  size_t slicePitchg4 = pitchg4 * N;
  
  char* sliceg4 = devPtrg4 + iz * slicePitchg4;
  float* g4 = (float*)(sliceg4 + iy * pitchg4);
  
  char* devPtrg4new = (char*) devpitchg4new.ptr;
  size_t pitchg4new = devpitchg4new.pitch;
  size_t slicePitchg4new = pitchg4new * N;
  
  char* sliceg4new = devPtrg4new + iz * slicePitchg4new;
  float* g4new = (float*)(sliceg4new + iy * pitchg4new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf5 = (char*) devpitchf5.ptr;
  size_t pitchf5 = devpitchf5.pitch;
  size_t slicePitchf5 = pitchf5 * N;
  
  char* slicef5 = devPtrf5 + iz * slicePitchf5;
  float* f5 = (float*)(slicef5 + iy * pitchf5);
  
  char* devPtrf5new = (char*) devpitchf5new.ptr;
  size_t pitchf5new = devpitchf5new.pitch;
  size_t slicePitchf5new = pitchf5new * N;
  
  char* slicef5new = devPtrf5new + iz * slicePitchf5new;
  float* f5new = (float*)(slicef5new + iy * pitchf5new);
  //---------------------------------------------------
  char* devPtrg5 = (char*) devpitchg5.ptr;
  size_t pitchg5 = devpitchg5.pitch;
  size_t slicePitchg5 = pitchg5 * N;
  
  char* sliceg5 = devPtrg5 + iz * slicePitchg5;
  float* g5 = (float*)(sliceg5 + iy * pitchg5);
  
  char* devPtrg5new = (char*) devpitchg5new.ptr;
  size_t pitchg5new = devpitchg5new.pitch;
  size_t slicePitchg5new = pitchg5new * N;
  
  char* sliceg5new = devPtrg5new + iz * slicePitchg5new;
  float* g5new = (float*)(sliceg5new + iy * pitchg5new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf6 = (char*) devpitchf6.ptr;
  size_t pitchf6 = devpitchf6.pitch;
  size_t slicePitchf6 = pitchf6 * N;
  
  char* slicef6 = devPtrf6 + iz * slicePitchf6;
  float* f6 = (float*)(slicef6 + iy * pitchf6);
  
  char* devPtrf6new = (char*) devpitchf6new.ptr;
  size_t pitchf6new = devpitchf6new.pitch;
  size_t slicePitchf6new = pitchf6new * N;
  
  char* slicef6new = devPtrf6new + iz * slicePitchf6new;
  float* f6new = (float*)(slicef6new + iy * pitchf6new);
  //---------------------------------------------------
  char* devPtrg6 = (char*) devpitchg6.ptr;
  size_t pitchg6 = devpitchg6.pitch;
  size_t slicePitchg6 = pitchg6 * N;
  
  char* sliceg6 = devPtrg6 + iz * slicePitchg6;
  float* g6 = (float*)(sliceg6 + iy * pitchg6);
  
  char* devPtrg6new = (char*) devpitchg6new.ptr;
  size_t pitchg6new = devpitchg6new.pitch;
  size_t slicePitchg6new = pitchg6new * N;
  
  char* sliceg6new = devPtrg6new + iz * slicePitchg6new;
  float* g6new = (float*)(sliceg6new + iy * pitchg6new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf7 = (char*) devpitchf7.ptr;
  size_t pitchf7 = devpitchf7.pitch;
  size_t slicePitchf7 = pitchf7 * N;
  
  char* slicef7 = devPtrf7 + iz * slicePitchf7;
  float* f7 = (float*)(slicef7 + iy * pitchf7);
  
  char* devPtrf7new = (char*) devpitchf7new.ptr;
  size_t pitchf7new = devpitchf7new.pitch;
  size_t slicePitchf7new = pitchf7new * N;
  
  char* slicef7new = devPtrf7new + iz * slicePitchf7new;
  float* f7new = (float*)(slicef7new + iy * pitchf7new);
  //---------------------------------------------------
  char* devPtrg7 = (char*) devpitchg7.ptr;
  size_t pitchg7 = devpitchg7.pitch;
  size_t slicePitchg7 = pitchg7 * N;
  
  char* sliceg7 = devPtrg7 + iz * slicePitchg7;
  float* g7 = (float*)(sliceg7 + iy * pitchg7);
  
  char* devPtrg7new = (char*) devpitchg7new.ptr;
  size_t pitchg7new = devpitchg7new.pitch;
  size_t slicePitchg7new = pitchg7new * N;
  
  char* sliceg7new = devPtrg7new + iz * slicePitchg7new;
  float* g7new = (float*)(sliceg7new + iy * pitchg7new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf8 = (char*) devpitchf8.ptr;
  size_t pitchf8 = devpitchf8.pitch;
  size_t slicePitchf8 = pitchf8 * N;
  
  char* slicef8 = devPtrf8 + iz * slicePitchf8;
  float* f8 = (float*)(slicef8 + iy * pitchf8);
  
  char* devPtrf8new = (char*) devpitchf8new.ptr;
  size_t pitchf8new = devpitchf8new.pitch;
  size_t slicePitchf8new = pitchf8new * N;
  
  char* slicef8new = devPtrf8new + iz * slicePitchf8new;
  float* f8new = (float*)(slicef8new + iy * pitchf8new);
  //---------------------------------------------------
  char* devPtrg8 = (char*) devpitchg8.ptr;
  size_t pitchg8 = devpitchg8.pitch;
  size_t slicePitchg8 = pitchg8 * N;
  
  char* sliceg8 = devPtrg8 + iz * slicePitchg8;
  float* g8 = (float*)(sliceg8 + iy * pitchg8);
  
  char* devPtrg8new = (char*) devpitchg8new.ptr;
  size_t pitchg8new = devpitchg8new.pitch;
  size_t slicePitchg8new = pitchg8new * N;
  
  char* sliceg8new = devPtrg8new + iz * slicePitchg8new;
  float* g8new = (float*)(sliceg8new + iy * pitchg8new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf9 = (char*) devpitchf9.ptr;
  size_t pitchf9 = devpitchf9.pitch;
  size_t slicePitchf9 = pitchf9 * N;
  
  char* slicef9 = devPtrf9 + iz * slicePitchf9;
  float* f9 = (float*)(slicef9 + iy * pitchf9);
  
  char* devPtrf9new = (char*) devpitchf9new.ptr;
  size_t pitchf9new = devpitchf9new.pitch;
  size_t slicePitchf9new = pitchf9new * N;
  
  char* slicef9new = devPtrf9new + iz * slicePitchf9new;
  float* f9new = (float*)(slicef9new + iy * pitchf9new);
  //---------------------------------------------------
  char* devPtrg9 = (char*) devpitchg9.ptr;
  size_t pitchg9 = devpitchg9.pitch;
  size_t slicePitchg9 = pitchg9 * N;
  
  char* sliceg9 = devPtrg9 + iz * slicePitchg9;
  float* g9 = (float*)(sliceg9 + iy * pitchg9);
  
  char* devPtrg9new = (char*) devpitchg9new.ptr;
  size_t pitchg9new = devpitchg9new.pitch;
  size_t slicePitchg9new = pitchg9new * N;
  
  char* sliceg9new = devPtrg9new + iz * slicePitchg9new;
  float* g9new = (float*)(sliceg9new + iy * pitchg9new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf10 = (char*) devpitchf10.ptr;
  size_t pitchf10 = devpitchf10.pitch;
  size_t slicePitchf10 = pitchf10 * N;
  
  char* slicef10 = devPtrf10 + iz * slicePitchf10;
  float* f10 = (float*)(slicef10 + iy * pitchf10);
  
  char* devPtrf10new = (char*) devpitchf10new.ptr;
  size_t pitchf10new = devpitchf10new.pitch;
  size_t slicePitchf10new = pitchf10new * N;
  
  char* slicef10new = devPtrf10new + iz * slicePitchf10new;
  float* f10new = (float*)(slicef10new + iy * pitchf10new);
  //---------------------------------------------------
  char* devPtrg10 = (char*) devpitchg10.ptr;
  size_t pitchg10 = devpitchg10.pitch;
  size_t slicePitchg10 = pitchg10 * N;
  
  char* sliceg10 = devPtrg10 + iz * slicePitchg10;
  float* g10 = (float*)(sliceg10 + iy * pitchg10);
  
  char* devPtrg10new = (char*) devpitchg10new.ptr;
  size_t pitchg10new = devpitchg10new.pitch;
  size_t slicePitchg10new = pitchg10new * N;
  
  char* sliceg10new = devPtrg10new + iz * slicePitchg10new;
  float* g10new = (float*)(sliceg10new + iy * pitchg10new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf11 = (char*) devpitchf11.ptr;
  size_t pitchf11 = devpitchf11.pitch;
  size_t slicePitchf11 = pitchf11 * N;
  
  char* slicef11 = devPtrf11 + iz * slicePitchf11;
  float* f11 = (float*)(slicef11 + iy * pitchf11);
  
  char* devPtrf11new = (char*) devpitchf11new.ptr;
  size_t pitchf11new = devpitchf11new.pitch;
  size_t slicePitchf11new = pitchf11new * N;
  
  char* slicef11new = devPtrf11new + iz * slicePitchf11new;
  float* f11new = (float*)(slicef11new + iy * pitchf11new);
  //---------------------------------------------------
  char* devPtrg11 = (char*) devpitchg11.ptr;
  size_t pitchg11 = devpitchg11.pitch;
  size_t slicePitchg11 = pitchg11 * N;
  
  char* sliceg11 = devPtrg11 + iz * slicePitchg11;
  float* g11 = (float*)(sliceg11 + iy * pitchg11);
  
  char* devPtrg11new = (char*) devpitchg11new.ptr;
  size_t pitchg11new = devpitchg11new.pitch;
  size_t slicePitchg11new = pitchg11new * N;
  
  char* sliceg11new = devPtrg11new + iz * slicePitchg11new;
  float* g11new = (float*)(sliceg11new + iy * pitchg11new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf12 = (char*) devpitchf12.ptr;
  size_t pitchf12 = devpitchf12.pitch;
  size_t slicePitchf12 = pitchf12 * N;
  
  char* slicef12 = devPtrf12 + iz * slicePitchf12;
  float* f12 = (float*)(slicef12 + iy * pitchf12);
  
  char* devPtrf12new = (char*) devpitchf12new.ptr;
  size_t pitchf12new = devpitchf12new.pitch;
  size_t slicePitchf12new = pitchf12new * N;
  
  char* slicef12new = devPtrf12new + iz * slicePitchf12new;
  float* f12new = (float*)(slicef12new + iy * pitchf12new);
  //---------------------------------------------------
  char* devPtrg12 = (char*) devpitchg12.ptr;
  size_t pitchg12 = devpitchg12.pitch;
  size_t slicePitchg12 = pitchg12 * N;
  
  char* sliceg12 = devPtrg12 + iz * slicePitchg12;
  float* g12 = (float*)(sliceg12 + iy * pitchg12);
  
  char* devPtrg12new = (char*) devpitchg12new.ptr;
  size_t pitchg12new = devpitchg12new.pitch;
  size_t slicePitchg12new = pitchg12new * N;
  
  char* sliceg12new = devPtrg12new + iz * slicePitchg12new;
  float* g12new = (float*)(sliceg12new + iy * pitchg12new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf13 = (char*) devpitchf13.ptr;
  size_t pitchf13 = devpitchf13.pitch;
  size_t slicePitchf13 = pitchf13 * N;
  
  char* slicef13 = devPtrf13 + iz * slicePitchf13;
  float* f13 = (float*)(slicef13 + iy * pitchf13);
  
  char* devPtrf13new = (char*) devpitchf13new.ptr;
  size_t pitchf13new = devpitchf13new.pitch;
  size_t slicePitchf13new = pitchf13new * N;
  
  char* slicef13new = devPtrf13new + iz * slicePitchf13new;
  float* f13new = (float*)(slicef13new + iy * pitchf13new);
  //---------------------------------------------------
  char* devPtrg13 = (char*) devpitchg13.ptr;
  size_t pitchg13 = devpitchg13.pitch;
  size_t slicePitchg13 = pitchg13 * N;
  
  char* sliceg13 = devPtrg13 + iz * slicePitchg13;
  float* g13 = (float*)(sliceg13 + iy * pitchg13);
  
  char* devPtrg13new = (char*) devpitchg13new.ptr;
  size_t pitchg13new = devpitchg13new.pitch;
  size_t slicePitchg13new = pitchg13new * N;
  
  char* sliceg13new = devPtrg13new + iz * slicePitchg13new;
  float* g13new = (float*)(sliceg13new + iy * pitchg13new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf14 = (char*) devpitchf14.ptr;
  size_t pitchf14 = devpitchf14.pitch;
  size_t slicePitchf14 = pitchf14 * N;
  
  char* slicef14 = devPtrf14 + iz * slicePitchf14;
  float* f14 = (float*)(slicef14 + iy * pitchf14);
  
  char* devPtrf14new = (char*) devpitchf14new.ptr;
  size_t pitchf14new = devpitchf14new.pitch;
  size_t slicePitchf14new = pitchf14new * N;
  
  char* slicef14new = devPtrf14new + iz * slicePitchf14new;
  float* f14new = (float*)(slicef14new + iy * pitchf14new);
  //---------------------------------------------------
  char* devPtrg14 = (char*) devpitchg14.ptr;
  size_t pitchg14 = devpitchg14.pitch;
  size_t slicePitchg14 = pitchg14 * N;
  
  char* sliceg14 = devPtrg14 + iz * slicePitchg14;
  float* g14 = (float*)(sliceg14 + iy * pitchg14);
  
  char* devPtrg14new = (char*) devpitchg14new.ptr;
  size_t pitchg14new = devpitchg14new.pitch;
  size_t slicePitchg14new = pitchg14new * N;
  
  char* sliceg14new = devPtrg14new + iz * slicePitchg14new;
  float* g14new = (float*)(sliceg14new + iy * pitchg14new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf15 = (char*) devpitchf15.ptr;
  size_t pitchf15 = devpitchf15.pitch;
  size_t slicePitchf15 = pitchf15 * N;
  
  char* slicef15 = devPtrf15 + iz * slicePitchf15;
  float* f15 = (float*)(slicef15 + iy * pitchf15);
  
  char* devPtrf15new = (char*) devpitchf15new.ptr;
  size_t pitchf15new = devpitchf15new.pitch;
  size_t slicePitchf15new = pitchf15new * N;
  
  char* slicef15new = devPtrf15new + iz * slicePitchf15new;
  float* f15new = (float*)(slicef15new + iy * pitchf15new);
  //---------------------------------------------------
  char* devPtrg15 = (char*) devpitchg15.ptr;
  size_t pitchg15 = devpitchg15.pitch;
  size_t slicePitchg15 = pitchg15 * N;
  
  char* sliceg15 = devPtrg15 + iz * slicePitchg15;
  float* g15 = (float*)(sliceg15 + iy * pitchg15);
  
  char* devPtrg15new = (char*) devpitchg15new.ptr;
  size_t pitchg15new = devpitchg15new.pitch;
  size_t slicePitchg15new = pitchg15new * N;
  
  char* sliceg15new = devPtrg15new + iz * slicePitchg15new;
  float* g15new = (float*)(sliceg15new + iy * pitchg15new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf16 = (char*) devpitchf16.ptr;
  size_t pitchf16 = devpitchf16.pitch;
  size_t slicePitchf16 = pitchf16 * N;
  
  char* slicef16 = devPtrf16 + iz * slicePitchf16;
  float* f16 = (float*)(slicef16 + iy * pitchf16);
  
  char* devPtrf16new = (char*) devpitchf16new.ptr;
  size_t pitchf16new = devpitchf16new.pitch;
  size_t slicePitchf16new = pitchf16new * N;
  
  char* slicef16new = devPtrf16new + iz * slicePitchf16new;
  float* f16new = (float*)(slicef16new + iy * pitchf16new);
  //---------------------------------------------------
  char* devPtrg16 = (char*) devpitchg16.ptr;
  size_t pitchg16 = devpitchg16.pitch;
  size_t slicePitchg16 = pitchg16 * N;
  
  char* sliceg16 = devPtrg16 + iz * slicePitchg16;
  float* g16 = (float*)(sliceg16 + iy * pitchg16);
  
  char* devPtrg16new = (char*) devpitchg16new.ptr;
  size_t pitchg16new = devpitchg16new.pitch;
  size_t slicePitchg16new = pitchg16new * N;
  
  char* sliceg16new = devPtrg16new + iz * slicePitchg16new;
  float* g16new = (float*)(sliceg16new + iy * pitchg16new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf17 = (char*) devpitchf17.ptr;
  size_t pitchf17 = devpitchf17.pitch;
  size_t slicePitchf17 = pitchf17 * N;
  
  char* slicef17 = devPtrf17 + iz * slicePitchf17;
  float* f17 = (float*)(slicef17 + iy * pitchf17);
  
  char* devPtrf17new = (char*) devpitchf17new.ptr;
  size_t pitchf17new = devpitchf17new.pitch;
  size_t slicePitchf17new = pitchf17new * N;
  
  char* slicef17new = devPtrf17new + iz * slicePitchf17new;
  float* f17new = (float*)(slicef17new + iy * pitchf17new);
  //---------------------------------------------------
  char* devPtrg17 = (char*) devpitchg17.ptr;
  size_t pitchg17 = devpitchg17.pitch;
  size_t slicePitchg17 = pitchg17 * N;
  
  char* sliceg17 = devPtrg17 + iz * slicePitchg17;
  float* g17 = (float*)(sliceg17 + iy * pitchg17);
  
  char* devPtrg17new = (char*) devpitchg17new.ptr;
  size_t pitchg17new = devpitchg17new.pitch;
  size_t slicePitchg17new = pitchg17new * N;
  
  char* sliceg17new = devPtrg17new + iz * slicePitchg17new;
  float* g17new = (float*)(sliceg17new + iy * pitchg17new);
  //-------------------------------------------------
  //--------------------------------------------------
  char* devPtrf18 = (char*) devpitchf18.ptr;
  size_t pitchf18 = devpitchf18.pitch;
  size_t slicePitchf18 = pitchf18 * N;
  
  char* slicef18 = devPtrf18 + iz * slicePitchf18;
  float* f18 = (float*)(slicef18 + iy * pitchf18);
  
  char* devPtrf18new = (char*) devpitchf18new.ptr;
  size_t pitchf18new = devpitchf18new.pitch;
  size_t slicePitchf18new = pitchf18new * N;
  
  char* slicef18new = devPtrf18new + iz * slicePitchf18new;
  float* f18new = (float*)(slicef18new + iy * pitchf18new);
  //---------------------------------------------------
  char* devPtrg18 = (char*) devpitchg18.ptr;
  size_t pitchg18 = devpitchg18.pitch;
  size_t slicePitchg18 = pitchg18 * N;
  
  char* sliceg18 = devPtrg18 + iz * slicePitchg18;
  float* g18 = (float*)(sliceg18 + iy * pitchg18);
  
  char* devPtrg18new = (char*) devpitchg18new.ptr;
  size_t pitchg18new = devpitchg18new.pitch;
  size_t slicePitchg18new = pitchg18new * N;
  
  char* sliceg18new = devPtrg18new + iz * slicePitchg18new;
  float* g18new = (float*)(sliceg18new + iy * pitchg18new);
  //-------------------------------------------------
  //--------------------------------------------------
  //printf("|%f",f0[390]);
  float Ux0,Uy0,Uz0,n0,rho0,P0,T0;

  
  if( ((ix - d_cm_x)*(ix - d_cm_x) + (iy - d_cm_y)*(iy - d_cm_y) + (iz - d_cm_z)*(iz - d_cm_z)) <= d_R*d_R){
  Ux0 = 0; 
  Uy0 = 0;
  Uz0 = 0;

  T0 = 0.0314;

  P0 = 0;
  n0 = P0/T0;
  rho0 = 3*n0*T0;

  }else{

  Ux0 = d_Ux(g0[ix],g1[ix],g2[ix],g3[ix],g4[ix],g5[ix],g6[ix],g7[ix],g8[ix],g9[ix],g10[ix],g11[ix],g12[ix],g13[ix],g14[ix],g15[ix],g16[ix],g17[ix],g18[ix]);
  Uy0 = d_Uy(g0[ix],g1[ix],g2[ix],g3[ix],g4[ix],g5[ix],g6[ix],g7[ix],g8[ix],g9[ix],g10[ix],g11[ix],g12[ix],g13[ix],g14[ix],g15[ix],g16[ix],g17[ix],g18[ix]);
  Uz0 = d_Uz(g0[ix],g1[ix],g2[ix],g3[ix],g4[ix],g5[ix],g6[ix],g7[ix],g8[ix],g9[ix],g10[ix],g11[ix],g12[ix],g13[ix],g14[ix],g15[ix],g16[ix],g17[ix],g18[ix]);
  n0 = d_n(f0[ix],f1[ix],f2[ix],f3[ix],f4[ix],f5[ix],f6[ix],f7[ix],f8[ix],f9[ix],f10[ix],f11[ix],f12[ix],f13[ix],f14[ix],f15[ix],f16[ix],f17[ix],f18[ix],Ux0,Uy0,Uz0);
  rho0 = d_rho(g0[ix],g1[ix],g2[ix],g3[ix],g4[ix],g5[ix],g6[ix],g7[ix],g8[ix],g9[ix],g10[ix],g11[ix],g12[ix],g13[ix],g14[ix],g15[ix],g16[ix],g17[ix],g18[ix]);
  P0 = d_P(g0[ix],g1[ix],g2[ix],g3[ix],g4[ix],g5[ix],g6[ix],g7[ix],g8[ix],g9[ix],g10[ix],g11[ix],g12[ix],g13[ix],g14[ix],g15[ix],g16[ix],g17[ix],g18[ix]);
  }

  
  f0new[ix] = (1.-1./(d_tau))*f0[ix]+(1./d_tau)*d_feq(0,n0,Ux0,Uy0,Uz0);
  g0new[ix] = (1.-1./(d_tau))*g0[ix]+(1./d_tau)*d_geq(0,rho0,P0,Ux0,Uy0,Uz0);
  

  f1new[ix] = (1.-1./(d_tau))*f1[ix]+(1./d_tau)*d_feq(1,n0,Ux0,Uy0,Uz0);
  g1new[ix] = (1.-1./(d_tau))*g1[ix]+(1./d_tau)*d_geq(1,rho0,P0,Ux0,Uy0,Uz0);

  f2new[ix] = (1.-1./(d_tau))*f2[ix]+(1./d_tau)*d_feq(2,n0,Ux0,Uy0,Uz0);
  g2new[ix] = (1.-1./(d_tau))*g2[ix]+(1./d_tau)*d_geq(2,rho0,P0,Ux0,Uy0,Uz0);

  f3new[ix] = (1.-1./(d_tau))*f3[ix]+(1./d_tau)*d_feq(3,n0,Ux0,Uy0,Uz0);
  g3new[ix] = (1.-1./(d_tau))*g3[ix]+(1./d_tau)*d_geq(3,rho0,P0,Ux0,Uy0,Uz0);

  f4new[ix] = (1.-1./(d_tau))*f4[ix]+(1./d_tau)*d_feq(4,n0,Ux0,Uy0,Uz0);
  g4new[ix] = (1.-1./(d_tau))*g4[ix]+(1./d_tau)*d_geq(4,rho0,P0,Ux0,Uy0,Uz0);

  f5new[ix] = (1.-1./(d_tau))*f5[ix]+(1./d_tau)*d_feq(5,n0,Ux0,Uy0,Uz0);
  g5new[ix] = (1.-1./(d_tau))*g5[ix]+(1./d_tau)*d_geq(5,rho0,P0,Ux0,Uy0,Uz0);

  f6new[ix] = (1.-1./(d_tau))*f6[ix]+(1./d_tau)*d_feq(6,n0,Ux0,Uy0,Uz0);
  g6new[ix] = (1.-1./(d_tau))*g6[ix]+(1./d_tau)*d_geq(6,rho0,P0,Ux0,Uy0,Uz0);

  f7new[ix] = (1.-1./(d_tau))*f7[ix]+(1./d_tau)*d_feq(7,n0,Ux0,Uy0,Uz0);
  g7new[ix] = (1.-1./(d_tau))*g7[ix]+(1./d_tau)*d_geq(7,rho0,P0,Ux0,Uy0,Uz0);

  f8new[ix] = (1.-1./(d_tau))*f8[ix]+(1./d_tau)*d_feq(8,n0,Ux0,Uy0,Uz0);
  g8new[ix] = (1.-1./(d_tau))*g8[ix]+(1./d_tau)*d_geq(8,rho0,P0,Ux0,Uy0,Uz0);

  f9new[ix] = (1.-1./(d_tau))*f9[ix]+(1./d_tau)*d_feq(9,n0,Ux0,Uy0,Uz0);
  g9new[ix] = (1.-1./(d_tau))*g9[ix]+(1./d_tau)*d_geq(9,rho0,P0,Ux0,Uy0,Uz0);

  f10new[ix] = (1.-1./(d_tau))*f10[ix]+(1./d_tau)*d_feq(10,n0,Ux0,Uy0,Uz0);
  g10new[ix] = (1.-1./(d_tau))*g10[ix]+(1./d_tau)*d_geq(10,rho0,P0,Ux0,Uy0,Uz0);

  f11new[ix] = (1.-1./(d_tau))*f11[ix]+(1./d_tau)*d_feq(11,n0,Ux0,Uy0,Uz0);
  g11new[ix] = (1.-1./(d_tau))*g11[ix]+(1./d_tau)*d_geq(11,rho0,P0,Ux0,Uy0,Uz0);

  f12new[ix] = (1.-1./(d_tau))*f12[ix]+(1./d_tau)*d_feq(12,n0,Ux0,Uy0,Uz0);
  g12new[ix] = (1.-1./(d_tau))*g12[ix]+(1./d_tau)*d_geq(12,rho0,P0,Ux0,Uy0,Uz0);

  f13new[ix] = (1.-1./(d_tau))*f13[ix]+(1./d_tau)*d_feq(13,n0,Ux0,Uy0,Uz0);
  g13new[ix] = (1.-1./(d_tau))*g13[ix]+(1./d_tau)*d_geq(13,rho0,P0,Ux0,Uy0,Uz0);

  f14new[ix] = (1.-1./(d_tau))*f14[ix]+(1./d_tau)*d_feq(14,n0,Ux0,Uy0,Uz0);
  g14new[ix] = (1.-1./(d_tau))*g14[ix]+(1./d_tau)*d_geq(14,rho0,P0,Ux0,Uy0,Uz0);

  f15new[ix] = (1.-1./(d_tau))*f15[ix]+(1./d_tau)*d_feq(15,n0,Ux0,Uy0,Uz0);
  g15new[ix] = (1.-1./(d_tau))*g15[ix]+(1./d_tau)*d_geq(15,rho0,P0,Ux0,Uy0,Uz0);

  f16new[ix] = (1.-1./(d_tau))*f16[ix]+(1./d_tau)*d_feq(16,n0,Ux0,Uy0,Uz0);
  g16new[ix] = (1.-1./(d_tau))*g16[ix]+(1./d_tau)*d_geq(16,rho0,P0,Ux0,Uy0,Uz0);

  f17new[ix] = (1.-1./(d_tau))*f17[ix]+(1./d_tau)*d_feq(17,n0,Ux0,Uy0,Uz0);
  g17new[ix] = (1.-1./(d_tau))*g17[ix]+(1./d_tau)*d_geq(17,rho0,P0,Ux0,Uy0,Uz0);

  f18new[ix] = (1.-1./(d_tau))*f18[ix]+(1./d_tau)*d_feq(18,n0,Ux0,Uy0,Uz0);
  g18new[ix] = (1.-1./(d_tau))*g18[ix]+(1./d_tau)*d_geq(18,rho0,P0,Ux0,Uy0,Uz0); 

  /*if(ix == 400 || ix == 399){
	printf("|**%.9f, %i",g0new[ix],ix);
	printf("|**%.9f, %i",g1new[ix],ix);
	printf("|**%.9f, %i",g2new[ix],ix);
	printf("|**%.9f, %i",g3new[ix],ix);
	printf("|**%.9f, %i",g4new[ix],ix);
	printf("|**%.9f, %i",g5new[ix],ix);
	printf("|**%.9f, %i",g6new[ix],ix);
	printf("|**%.9f, %i",g7new[ix],ix);
  }*/
  

}
//--------------------Class-------------------------------
class LatticeBoltzmann{
private:
  float h_w[Q];
  int h_Vx[Q],h_Vy[Q],h_Vz[Q];
 
  
  hipPitchedPtr devPitchedf0;           hipPitchedPtr devPitchedf0new;           hipPitchedPtr devPitchedg0;           hipPitchedPtr devPitchedg0new;
  hipPitchedPtr devPitchedf1;           hipPitchedPtr devPitchedf1new;           hipPitchedPtr devPitchedg1;           hipPitchedPtr devPitchedg1new;
  hipPitchedPtr devPitchedf2;           hipPitchedPtr devPitchedf2new;           hipPitchedPtr devPitchedg2;           hipPitchedPtr devPitchedg2new;
  hipPitchedPtr devPitchedf3;           hipPitchedPtr devPitchedf3new;           hipPitchedPtr devPitchedg3;           hipPitchedPtr devPitchedg3new;
  hipPitchedPtr devPitchedf4;           hipPitchedPtr devPitchedf4new;           hipPitchedPtr devPitchedg4;           hipPitchedPtr devPitchedg4new;
  hipPitchedPtr devPitchedf5;           hipPitchedPtr devPitchedf5new;           hipPitchedPtr devPitchedg5;           hipPitchedPtr devPitchedg5new;
  hipPitchedPtr devPitchedf6;           hipPitchedPtr devPitchedf6new;           hipPitchedPtr devPitchedg6;           hipPitchedPtr devPitchedg6new;
  hipPitchedPtr devPitchedf7;           hipPitchedPtr devPitchedf7new;           hipPitchedPtr devPitchedg7;           hipPitchedPtr devPitchedg7new;
  hipPitchedPtr devPitchedf8;           hipPitchedPtr devPitchedf8new;           hipPitchedPtr devPitchedg8;           hipPitchedPtr devPitchedg8new;
  hipPitchedPtr devPitchedf9;           hipPitchedPtr devPitchedf9new;           hipPitchedPtr devPitchedg9;           hipPitchedPtr devPitchedg9new;
  hipPitchedPtr devPitchedf10;           hipPitchedPtr devPitchedf10new;           hipPitchedPtr devPitchedg10;           hipPitchedPtr devPitchedg10new;
  hipPitchedPtr devPitchedf11;           hipPitchedPtr devPitchedf11new;           hipPitchedPtr devPitchedg11;           hipPitchedPtr devPitchedg11new;
  hipPitchedPtr devPitchedf12;           hipPitchedPtr devPitchedf12new;           hipPitchedPtr devPitchedg12;           hipPitchedPtr devPitchedg12new;
  hipPitchedPtr devPitchedf13;           hipPitchedPtr devPitchedf13new;           hipPitchedPtr devPitchedg13;           hipPitchedPtr devPitchedg13new;
  hipPitchedPtr devPitchedf14;           hipPitchedPtr devPitchedf14new;           hipPitchedPtr devPitchedg14;           hipPitchedPtr devPitchedg14new;
  hipPitchedPtr devPitchedf15;           hipPitchedPtr devPitchedf15new;           hipPitchedPtr devPitchedg15;           hipPitchedPtr devPitchedg15new;
  hipPitchedPtr devPitchedf16;           hipPitchedPtr devPitchedf16new;           hipPitchedPtr devPitchedg16;           hipPitchedPtr devPitchedg16new;
  hipPitchedPtr devPitchedf17;           hipPitchedPtr devPitchedf17new;           hipPitchedPtr devPitchedg17;           hipPitchedPtr devPitchedg17new;
  hipPitchedPtr devPitchedf18;           hipPitchedPtr devPitchedf18new;           hipPitchedPtr devPitchedg18;           hipPitchedPtr devPitchedg18new;

  hipMemcpy3DParms p0 = { 0 };          hipMemcpy3DParms p0new = { 0 };            hipMemcpy3DParms q0 = { 0 };          hipMemcpy3DParms q0new = { 0 };
  hipMemcpy3DParms p1 = { 0 };          hipMemcpy3DParms p1new = { 0 };            hipMemcpy3DParms q1 = { 0 };          hipMemcpy3DParms q1new = { 0 };
  hipMemcpy3DParms p2 = { 0 };          hipMemcpy3DParms p2new = { 0 };            hipMemcpy3DParms q2 = { 0 };          hipMemcpy3DParms q2new = { 0 };
  hipMemcpy3DParms p3 = { 0 };          hipMemcpy3DParms p3new = { 0 };            hipMemcpy3DParms q3 = { 0 };          hipMemcpy3DParms q3new = { 0 };
  hipMemcpy3DParms p4 = { 0 };          hipMemcpy3DParms p4new = { 0 };            hipMemcpy3DParms q4 = { 0 };          hipMemcpy3DParms q4new = { 0 };
  hipMemcpy3DParms p5 = { 0 };          hipMemcpy3DParms p5new = { 0 };            hipMemcpy3DParms q5 = { 0 };          hipMemcpy3DParms q5new = { 0 };
  hipMemcpy3DParms p6 = { 0 };          hipMemcpy3DParms p6new = { 0 };            hipMemcpy3DParms q6 = { 0 };          hipMemcpy3DParms q6new = { 0 };
  hipMemcpy3DParms p7 = { 0 };          hipMemcpy3DParms p7new = { 0 };            hipMemcpy3DParms q7 = { 0 };          hipMemcpy3DParms q7new = { 0 };
  hipMemcpy3DParms p8 = { 0 };          hipMemcpy3DParms p8new = { 0 };            hipMemcpy3DParms q8 = { 0 };          hipMemcpy3DParms q8new = { 0 };
  hipMemcpy3DParms p9 = { 0 };          hipMemcpy3DParms p9new = { 0 };            hipMemcpy3DParms q9 = { 0 };          hipMemcpy3DParms q9new = { 0 };
  hipMemcpy3DParms p10 = { 0 };          hipMemcpy3DParms p10new = { 0 };            hipMemcpy3DParms q10 = { 0 };          hipMemcpy3DParms q10new = { 0 };
  hipMemcpy3DParms p11 = { 0 };          hipMemcpy3DParms p11new = { 0 };            hipMemcpy3DParms q11 = { 0 };          hipMemcpy3DParms q11new = { 0 };
  hipMemcpy3DParms p12 = { 0 };          hipMemcpy3DParms p12new = { 0 };            hipMemcpy3DParms q12 = { 0 };          hipMemcpy3DParms q12new = { 0 };
  hipMemcpy3DParms p13 = { 0 };          hipMemcpy3DParms p13new = { 0 };            hipMemcpy3DParms q13 = { 0 };          hipMemcpy3DParms q13new = { 0 };
  hipMemcpy3DParms p14 = { 0 };          hipMemcpy3DParms p14new = { 0 };            hipMemcpy3DParms q14 = { 0 };          hipMemcpy3DParms q14new = { 0 };
  hipMemcpy3DParms p15 = { 0 };          hipMemcpy3DParms p15new = { 0 };            hipMemcpy3DParms q15 = { 0 };          hipMemcpy3DParms q15new = { 0 };
  hipMemcpy3DParms p16 = { 0 };          hipMemcpy3DParms p16new = { 0 };            hipMemcpy3DParms q16 = { 0 };          hipMemcpy3DParms q16new = { 0 };
  hipMemcpy3DParms p17 = { 0 };          hipMemcpy3DParms p17new = { 0 };            hipMemcpy3DParms q17 = { 0 };          hipMemcpy3DParms q17new = { 0 };
  hipMemcpy3DParms p18 = { 0 };          hipMemcpy3DParms p18new = { 0 };            hipMemcpy3DParms q18 = { 0 };          hipMemcpy3DParms q18new = { 0 };
 
  float h_f0[W][N][M]; float h_f0new[W][N][M];  float h_g0[W][N][M];   float h_g0new[W][N][M];
  float h_f1[W][N][M]; float h_f1new[W][N][M];  float h_g1[W][N][M];   float h_g1new[W][N][M];
  float h_f2[W][N][M]; float h_f2new[W][N][M];  float h_g2[W][N][M];   float h_g2new[W][N][M];
  float h_f3[W][N][M]; float h_f3new[W][N][M];  float h_g3[W][N][M];   float h_g3new[W][N][M];
  float h_f4[W][N][M]; float h_f4new[W][N][M];  float h_g4[W][N][M];   float h_g4new[W][N][M];
  float h_f5[W][N][M]; float h_f5new[W][N][M];  float h_g5[W][N][M];   float h_g5new[W][N][M];
  float h_f6[W][N][M]; float h_f6new[W][N][M];  float h_g6[W][N][M];   float h_g6new[W][N][M];
  float h_f7[W][N][M]; float h_f7new[W][N][M];  float h_g7[W][N][M];   float h_g7new[W][N][M];
  float h_f8[W][N][M]; float h_f8new[W][N][M];  float h_g8[W][N][M];   float h_g8new[W][N][M];
  float h_f9[W][N][M]; float h_f9new[W][N][M];  float h_g9[W][N][M];   float h_g9new[W][N][M];
  float h_f10[W][N][M]; float h_f10new[W][N][M];  float h_g10[W][N][M];   float h_g10new[W][N][M];
  float h_f11[W][N][M]; float h_f11new[W][N][M];  float h_g11[W][N][M];   float h_g11new[W][N][M];
  float h_f12[W][N][M]; float h_f12new[W][N][M];  float h_g12[W][N][M];   float h_g12new[W][N][M];
  float h_f13[W][N][M]; float h_f13new[W][N][M];  float h_g13[W][N][M];   float h_g13new[W][N][M];
  float h_f14[W][N][M]; float h_f14new[W][N][M];  float h_g14[W][N][M];   float h_g14new[W][N][M];
  float h_f15[W][N][M]; float h_f15new[W][N][M];  float h_g15[W][N][M];   float h_g15new[W][N][M];
  float h_f16[W][N][M]; float h_f16new[W][N][M];  float h_g16[W][N][M];   float h_g16new[W][N][M];
  float h_f17[W][N][M]; float h_f17new[W][N][M];  float h_g17[W][N][M];   float h_g17new[W][N][M];
  float h_f18[W][N][M]; float h_f18new[W][N][M];  float h_g18[W][N][M];   float h_g18new[W][N][M];
  
public:
  LatticeBoltzmann(void);
  ~LatticeBoltzmann(void);
  void Start(float Ux0,float Uy0,float Uz0,float rho0, float rho1, float n0, float n1,float P0,float P1);
  void Advection(void);
  void Collision(void);
  void Show(void);
  float h_Ux(int ix,int iy,int iz);
  float h_Uy(int ix,int iy,int iz);
  float h_Uz(int ix,int iy,int iz);
  float h_gamma(float Ux0,float Uy0,float Uz0);
  float h_n(int ix,int iy,int iz,float Ux0,float Uy0,float Uz0);
  float h_P(int ix,int iy,int iz);
  float h_rho(int ix,int iy,int iz);
  float h_feq(int i,float n0,float Ux0,float Uy0,float Uz0);
  float h_geq(int i,float rho0,float P0,float Ux0,float Uy0,float Uz0);
  void Print(const char * NombreArchivo);
};

LatticeBoltzmann::LatticeBoltzmann(void){
  // --- 3D pitched allocation and host->device memcopy
  hipExtent extent = make_hipExtent(M * sizeof(float), N, W);
  hipMalloc3D(&devPitchedf0, extent);   hipMalloc3D(&devPitchedf0new, extent);   hipMalloc3D(&devPitchedg0, extent);   hipMalloc3D(&devPitchedg0new, extent);
  hipMalloc3D(&devPitchedf1, extent);   hipMalloc3D(&devPitchedf1new, extent);   hipMalloc3D(&devPitchedg1, extent);   hipMalloc3D(&devPitchedg1new, extent);
  hipMalloc3D(&devPitchedf2, extent);   hipMalloc3D(&devPitchedf2new, extent);   hipMalloc3D(&devPitchedg2, extent);   hipMalloc3D(&devPitchedg2new, extent);
  hipMalloc3D(&devPitchedf3, extent);   hipMalloc3D(&devPitchedf3new, extent);   hipMalloc3D(&devPitchedg3, extent);   hipMalloc3D(&devPitchedg3new, extent);
  hipMalloc3D(&devPitchedf4, extent);   hipMalloc3D(&devPitchedf4new, extent);   hipMalloc3D(&devPitchedg4, extent);   hipMalloc3D(&devPitchedg4new, extent);
  hipMalloc3D(&devPitchedf5, extent);   hipMalloc3D(&devPitchedf5new, extent);   hipMalloc3D(&devPitchedg5, extent);   hipMalloc3D(&devPitchedg5new, extent);
  hipMalloc3D(&devPitchedf6, extent);   hipMalloc3D(&devPitchedf6new, extent);   hipMalloc3D(&devPitchedg6, extent);   hipMalloc3D(&devPitchedg6new, extent);
  hipMalloc3D(&devPitchedf7, extent);   hipMalloc3D(&devPitchedf7new, extent);   hipMalloc3D(&devPitchedg7, extent);   hipMalloc3D(&devPitchedg7new, extent);
  hipMalloc3D(&devPitchedf8, extent);   hipMalloc3D(&devPitchedf8new, extent);   hipMalloc3D(&devPitchedg8, extent);   hipMalloc3D(&devPitchedg8new, extent);
  hipMalloc3D(&devPitchedf9, extent);   hipMalloc3D(&devPitchedf9new, extent);   hipMalloc3D(&devPitchedg9, extent);   hipMalloc3D(&devPitchedg9new, extent);
  hipMalloc3D(&devPitchedf10, extent);   hipMalloc3D(&devPitchedf10new, extent);   hipMalloc3D(&devPitchedg10, extent);   hipMalloc3D(&devPitchedg10new, extent);
  hipMalloc3D(&devPitchedf11, extent);   hipMalloc3D(&devPitchedf11new, extent);   hipMalloc3D(&devPitchedg11, extent);   hipMalloc3D(&devPitchedg11new, extent);
  hipMalloc3D(&devPitchedf12, extent);   hipMalloc3D(&devPitchedf12new, extent);   hipMalloc3D(&devPitchedg12, extent);   hipMalloc3D(&devPitchedg12new, extent);
  hipMalloc3D(&devPitchedf13, extent);   hipMalloc3D(&devPitchedf13new, extent);   hipMalloc3D(&devPitchedg13, extent);   hipMalloc3D(&devPitchedg13new, extent);
  hipMalloc3D(&devPitchedf14, extent);   hipMalloc3D(&devPitchedf14new, extent);   hipMalloc3D(&devPitchedg14, extent);   hipMalloc3D(&devPitchedg14new, extent);
  hipMalloc3D(&devPitchedf15, extent);   hipMalloc3D(&devPitchedf15new, extent);   hipMalloc3D(&devPitchedg15, extent);   hipMalloc3D(&devPitchedg15new, extent);
  hipMalloc3D(&devPitchedf16, extent);   hipMalloc3D(&devPitchedf16new, extent);   hipMalloc3D(&devPitchedg16, extent);   hipMalloc3D(&devPitchedg16new, extent);
  hipMalloc3D(&devPitchedf17, extent);   hipMalloc3D(&devPitchedf17new, extent);   hipMalloc3D(&devPitchedg17, extent);   hipMalloc3D(&devPitchedg17new, extent);
  hipMalloc3D(&devPitchedf18, extent);   hipMalloc3D(&devPitchedf18new, extent);   hipMalloc3D(&devPitchedg18, extent);   hipMalloc3D(&devPitchedg18new, extent);
}
LatticeBoltzmann::~LatticeBoltzmann(void){
  //Free memory on device
  hipFree(&devPitchedf0.ptr);   hipFree(&devPitchedf0new.ptr);   hipFree(&devPitchedg0.ptr);   hipFree(&devPitchedg0new.ptr);
  hipFree(&devPitchedf1.ptr);   hipFree(&devPitchedf1new.ptr);   hipFree(&devPitchedg1.ptr);   hipFree(&devPitchedg1new.ptr);
  hipFree(&devPitchedf2.ptr);   hipFree(&devPitchedf2new.ptr);   hipFree(&devPitchedg2.ptr);   hipFree(&devPitchedg2new.ptr);
  hipFree(&devPitchedf3.ptr);   hipFree(&devPitchedf3new.ptr);   hipFree(&devPitchedg3.ptr);   hipFree(&devPitchedg3new.ptr);
  hipFree(&devPitchedf4.ptr);   hipFree(&devPitchedf4new.ptr);   hipFree(&devPitchedg4.ptr);   hipFree(&devPitchedg4new.ptr);
  hipFree(&devPitchedf5.ptr);   hipFree(&devPitchedf5new.ptr);   hipFree(&devPitchedg5.ptr);   hipFree(&devPitchedg5new.ptr);
  hipFree(&devPitchedf6.ptr);   hipFree(&devPitchedf6new.ptr);   hipFree(&devPitchedg6.ptr);   hipFree(&devPitchedg6new.ptr);
  hipFree(&devPitchedf7.ptr);   hipFree(&devPitchedf7new.ptr);   hipFree(&devPitchedg7.ptr);   hipFree(&devPitchedg7new.ptr);
  hipFree(&devPitchedf8.ptr);   hipFree(&devPitchedf8new.ptr);   hipFree(&devPitchedg8.ptr);   hipFree(&devPitchedg8new.ptr);
  hipFree(&devPitchedf9.ptr);   hipFree(&devPitchedf9new.ptr);   hipFree(&devPitchedg9.ptr);   hipFree(&devPitchedg9new.ptr);
  hipFree(&devPitchedf10.ptr);   hipFree(&devPitchedf10new.ptr);   hipFree(&devPitchedg10.ptr);   hipFree(&devPitchedg10new.ptr);
  hipFree(&devPitchedf11.ptr);   hipFree(&devPitchedf11new.ptr);   hipFree(&devPitchedg11.ptr);   hipFree(&devPitchedg11new.ptr);
  hipFree(&devPitchedf12.ptr);   hipFree(&devPitchedf12new.ptr);   hipFree(&devPitchedg12.ptr);   hipFree(&devPitchedg12new.ptr);
  hipFree(&devPitchedf13.ptr);   hipFree(&devPitchedf13new.ptr);   hipFree(&devPitchedg13.ptr);   hipFree(&devPitchedg13new.ptr);
  hipFree(&devPitchedf14.ptr);   hipFree(&devPitchedf14new.ptr);   hipFree(&devPitchedg14.ptr);   hipFree(&devPitchedg14new.ptr);
  hipFree(&devPitchedf15.ptr);   hipFree(&devPitchedf15new.ptr);   hipFree(&devPitchedg15.ptr);   hipFree(&devPitchedg15new.ptr);
  hipFree(&devPitchedf16.ptr);   hipFree(&devPitchedf16new.ptr);   hipFree(&devPitchedg16.ptr);   hipFree(&devPitchedg16new.ptr);
  hipFree(&devPitchedf17.ptr);   hipFree(&devPitchedf17new.ptr);   hipFree(&devPitchedg17.ptr);   hipFree(&devPitchedg17new.ptr);
  hipFree(&devPitchedf18.ptr);   hipFree(&devPitchedf18new.ptr);   hipFree(&devPitchedg18.ptr);   hipFree(&devPitchedg18new.ptr);
}
void LatticeBoltzmann::Start(float Ux0,float Uy0,float Uz0,float rho0, float rho1, float n0, float n1,float P0,float P1){
  int i,j;
  int V[3][Q];
  //-----------Weights----------------
  h_w[0]=1./3.;
  for(i=1;i<7;i++)
    h_w[i]=1./18.;
  for(i=7;i<Q;i++)
    h_w[i]=1./36.;
  //-----------Velocities-------------
  for(i=0;i<Q;i++){
    for(j=0;j<3;j++){V[j][i] = 0;}
  }
  int counter = 0;
  for(i=1;i<7;i++){
    V[counter][i]=pow(-1,i+1);
    if(i%2==0){counter = counter+1;}
  }
  int counter1 = 0;
  int counter2 = 0;
  int counter3 = 0;
  for(i=7;i<Q;i++){
    if(i<15){
      V[counter3][i] = pow(-1,counter2);
      V[(counter1%2)+counter3+1][i] = pow(-1,(i+1)%2);
    }else{
      V[counter3][i] = pow(-1,counter1);
      V[counter3+1][i] = pow(-1,(i+1)%2);
    }
    if((i-6)%2==0){counter1 = counter1 + 1;}
    if((i-6)%4==0){counter2 = counter2 + 1;}
    if((i-6)%8==0){counter3 = counter3 + 1;}
  }
  for(i = 0; i < Q; i++){
    h_Vx[i] = V[0][i];
    h_Vy[i] = V[1][i];
    h_Vz[i] = V[2][i];
  }

  //------Enviarlas al Device-----------------
  hipMemcpyToSymbol(HIP_SYMBOL(d_w),h_w,Q*sizeof(float),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vx),h_Vx,Q*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vy),h_Vy,Q*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Vz),h_Vz,Q*sizeof(int),0,hipMemcpyHostToDevice);
  //FUNCIONES DE DISTRIBUCION
  int ix,iy,iz;
  float P,n,rho;
  //Cargar valores en el Host
  for(ix=0;ix<M;ix++)
    for(iy=0;iy<N;iy++)
      for(iz=0;iz<W;iz++){
        h_f0new[iz][iy][ix] = 0;    h_g0new[iz][iy][ix] = 0;
        h_f1new[iz][iy][ix] = 0;    h_g1new[iz][iy][ix] = 0;
        h_f2new[iz][iy][ix] = 0;    h_g2new[iz][iy][ix] = 0;
        h_f3new[iz][iy][ix] = 0;    h_g3new[iz][iy][ix] = 0;
        h_f4new[iz][iy][ix] = 0;    h_g4new[iz][iy][ix] = 0;
        h_f5new[iz][iy][ix] = 0;    h_g5new[iz][iy][ix] = 0;
        h_f6new[iz][iy][ix] = 0;    h_g6new[iz][iy][ix] = 0;
        h_f7new[iz][iy][ix] = 0;    h_g7new[iz][iy][ix] = 0;
        h_f8new[iz][iy][ix] = 0;    h_g8new[iz][iy][ix] = 0;
        h_f9new[iz][iy][ix] = 0;    h_g9new[iz][iy][ix] = 0;
        h_f10new[iz][iy][ix] =0;    h_g10new[iz][iy][ix] = 0;
        h_f11new[iz][iy][ix] =0;    h_g11new[iz][iy][ix] = 0;
        h_f12new[iz][iy][ix] =0;    h_g12new[iz][iy][ix] = 0;
        h_f13new[iz][iy][ix] =0;    h_g13new[iz][iy][ix] = 0;
        h_f14new[iz][iy][ix] =0;    h_g14new[iz][iy][ix] = 0;
        h_f15new[iz][iy][ix] =0;    h_g15new[iz][iy][ix] = 0;
        h_f16new[iz][iy][ix] =0;    h_g16new[iz][iy][ix] = 0;
        h_f17new[iz][iy][ix] =0;    h_g17new[iz][iy][ix] = 0;
        h_f18new[iz][iy][ix] =0;    h_g18new[iz][iy][ix] = 0;
	//---------------------------
        if(ix < int(M*0.5)){
          P = P0;
          n = n0;
          rho = rho0;
        }else if(ix >= int(M*0.5)){
          P = P1;
          n = n1;
          rho = rho1;
        }
	//--------------------------
        /*
	P = (P1-P0)*atan(ix-400)*0.5+(P0+P1)*0.5;
        n = (n1-n0)*atan(ix-400)*0.5+(n0+n1)*0.5;
        rho = (rho1-rho0)*atan(ix-400)*0.5+(rho0+rho1)*0.5;
        */

        h_f0[iz][iy][ix] = h_feq(0,n,Ux0,Uy0,Uz0);    h_g0[iz][iy][ix] = h_geq(0,rho,P,Ux0,Uy0,Uz0);
        h_f1[iz][iy][ix] = h_feq(1,n,Ux0,Uy0,Uz0);    h_g1[iz][iy][ix] = h_geq(1,rho,P,Ux0,Uy0,Uz0);
        h_f2[iz][iy][ix] = h_feq(2,n,Ux0,Uy0,Uz0);    h_g2[iz][iy][ix] = h_geq(2,rho,P,Ux0,Uy0,Uz0);
        h_f3[iz][iy][ix] = h_feq(3,n,Ux0,Uy0,Uz0);    h_g3[iz][iy][ix] = h_geq(3,rho,P,Ux0,Uy0,Uz0);
        h_f4[iz][iy][ix] = h_feq(4,n,Ux0,Uy0,Uz0);    h_g4[iz][iy][ix] = h_geq(4,rho,P,Ux0,Uy0,Uz0);
        h_f5[iz][iy][ix] = h_feq(5,n,Ux0,Uy0,Uz0);    h_g5[iz][iy][ix] = h_geq(5,rho,P,Ux0,Uy0,Uz0);
        h_f6[iz][iy][ix] = h_feq(6,n,Ux0,Uy0,Uz0);    h_g6[iz][iy][ix] = h_geq(6,rho,P,Ux0,Uy0,Uz0);
        h_f7[iz][iy][ix] = h_feq(7,n,Ux0,Uy0,Uz0);    h_g7[iz][iy][ix] = h_geq(7,rho,P,Ux0,Uy0,Uz0);
        h_f8[iz][iy][ix] = h_feq(8,n,Ux0,Uy0,Uz0);    h_g8[iz][iy][ix] = h_geq(8,rho,P,Ux0,Uy0,Uz0);
        h_f9[iz][iy][ix] = h_feq(9,n,Ux0,Uy0,Uz0);    h_g9[iz][iy][ix] = h_geq(9,rho,P,Ux0,Uy0,Uz0);
        h_f10[iz][iy][ix] = h_feq(10,n,Ux0,Uy0,Uz0);    h_g10[iz][iy][ix] = h_geq(10,rho,P,Ux0,Uy0,Uz0);
        h_f11[iz][iy][ix] = h_feq(11,n,Ux0,Uy0,Uz0);    h_g11[iz][iy][ix] = h_geq(11,rho,P,Ux0,Uy0,Uz0);
        h_f12[iz][iy][ix] = h_feq(12,n,Ux0,Uy0,Uz0);    h_g12[iz][iy][ix] = h_geq(12,rho,P,Ux0,Uy0,Uz0);
        h_f13[iz][iy][ix] = h_feq(13,n,Ux0,Uy0,Uz0);    h_g13[iz][iy][ix] = h_geq(13,rho,P,Ux0,Uy0,Uz0);
        h_f14[iz][iy][ix] = h_feq(14,n,Ux0,Uy0,Uz0);    h_g14[iz][iy][ix] = h_geq(14,rho,P,Ux0,Uy0,Uz0);
        h_f15[iz][iy][ix] = h_feq(15,n,Ux0,Uy0,Uz0);    h_g15[iz][iy][ix] = h_geq(15,rho,P,Ux0,Uy0,Uz0);
        h_f16[iz][iy][ix] = h_feq(16,n,Ux0,Uy0,Uz0);    h_g16[iz][iy][ix] = h_geq(16,rho,P,Ux0,Uy0,Uz0);
        h_f17[iz][iy][ix] = h_feq(17,n,Ux0,Uy0,Uz0);    h_g17[iz][iy][ix] = h_geq(17,rho,P,Ux0,Uy0,Uz0);
        h_f18[iz][iy][ix] = h_feq(18,n,Ux0,Uy0,Uz0);    h_g18[iz][iy][ix] = h_geq(18,rho,P,Ux0,Uy0,Uz0);
      }
  //cout << h_g10[0][0][39] << endl;
  //Llevar al Devic
  p0.srcPtr.ptr = h_f0;
  p0.srcPtr.pitch = M * sizeof(float);
  p0.srcPtr.xsize = M;
  p0.srcPtr.ysize = N;
  p0.dstPtr.ptr = devPitchedf0.ptr;
  p0.dstPtr.pitch = devPitchedf0.pitch;
  p0.dstPtr.xsize = M;
  p0.dstPtr.ysize = N;
  p0.extent.width = M * sizeof(float);
  p0.extent.height = N;
  p0.extent.depth = W;
  p0.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p0);

  p1.srcPtr.ptr = h_f1;
  p1.srcPtr.pitch = M * sizeof(float);
  p1.srcPtr.xsize = M;
  p1.srcPtr.ysize = N;
  p1.dstPtr.ptr = devPitchedf1.ptr;
  p1.dstPtr.pitch = devPitchedf1.pitch;
  p1.dstPtr.xsize = M;
  p1.dstPtr.ysize = N;
  p1.extent.width = M * sizeof(float);
  p1.extent.height = N;
  p1.extent.depth = W;
  p1.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p1);

  p2.srcPtr.ptr = h_f2;
  p2.srcPtr.pitch = M * sizeof(float);
  p2.srcPtr.xsize = M;
  p2.srcPtr.ysize = N;
  p2.dstPtr.ptr = devPitchedf2.ptr;
  p2.dstPtr.pitch = devPitchedf2.pitch;
  p2.dstPtr.xsize = M;
  p2.dstPtr.ysize = N;
  p2.extent.width = M * sizeof(float);
  p2.extent.height = N;
  p2.extent.depth = W;
  p2.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p2);

  p3.srcPtr.ptr = h_f3;
  p3.srcPtr.pitch = M * sizeof(float);
  p3.srcPtr.xsize = M;
  p3.srcPtr.ysize = N;
  p3.dstPtr.ptr = devPitchedf3.ptr;
  p3.dstPtr.pitch = devPitchedf3.pitch;
  p3.dstPtr.xsize = M;
  p3.dstPtr.ysize = N;
  p3.extent.width = M * sizeof(float);
  p3.extent.height = N;
  p3.extent.depth = W;
  p3.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p3);

  p4.srcPtr.ptr = h_f4;
  p4.srcPtr.pitch = M * sizeof(float);
  p4.srcPtr.xsize = M;
  p4.srcPtr.ysize = N;
  p4.dstPtr.ptr = devPitchedf4.ptr;
  p4.dstPtr.pitch = devPitchedf4.pitch;
  p4.dstPtr.xsize = M;
  p4.dstPtr.ysize = N;
  p4.extent.width = M * sizeof(float);
  p4.extent.height = N;
  p4.extent.depth = W;
  p4.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p4);

  p5.srcPtr.ptr = h_f5;
  p5.srcPtr.pitch = M * sizeof(float);
  p5.srcPtr.xsize = M;
  p5.srcPtr.ysize = N;
  p5.dstPtr.ptr = devPitchedf5.ptr;
  p5.dstPtr.pitch = devPitchedf5.pitch;
  p5.dstPtr.xsize = M;
  p5.dstPtr.ysize = N;
  p5.extent.width = M * sizeof(float);
  p5.extent.height = N;
  p5.extent.depth = W;
  p5.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p5);

  p6.srcPtr.ptr = h_f6;
  p6.srcPtr.pitch = M * sizeof(float);
  p6.srcPtr.xsize = M;
  p6.srcPtr.ysize = N;
  p6.dstPtr.ptr = devPitchedf6.ptr;
  p6.dstPtr.pitch = devPitchedf6.pitch;
  p6.dstPtr.xsize = M;
  p6.dstPtr.ysize = N;
  p6.extent.width = M * sizeof(float);
  p6.extent.height = N;
  p6.extent.depth = W;
  p6.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p6);

  p7.srcPtr.ptr = h_f7;
  p7.srcPtr.pitch = M * sizeof(float);
  p7.srcPtr.xsize = M;
  p7.srcPtr.ysize = N;
  p7.dstPtr.ptr = devPitchedf7.ptr;
  p7.dstPtr.pitch = devPitchedf7.pitch;
  p7.dstPtr.xsize = M;
  p7.dstPtr.ysize = N;
  p7.extent.width = M * sizeof(float);
  p7.extent.height = N;
  p7.extent.depth = W;
  p7.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p7);

  p8.srcPtr.ptr = h_f8;
  p8.srcPtr.pitch = M * sizeof(float);
  p8.srcPtr.xsize = M;
  p8.srcPtr.ysize = N;
  p8.dstPtr.ptr = devPitchedf8.ptr;
  p8.dstPtr.pitch = devPitchedf8.pitch;
  p8.dstPtr.xsize = M;
  p8.dstPtr.ysize = N;
  p8.extent.width = M * sizeof(float);
  p8.extent.height = N;
  p8.extent.depth = W;
  p8.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p8);

  p9.srcPtr.ptr = h_f9;
  p9.srcPtr.pitch = M * sizeof(float);
  p9.srcPtr.xsize = M;
  p9.srcPtr.ysize = N;
  p9.dstPtr.ptr = devPitchedf9.ptr;
  p9.dstPtr.pitch = devPitchedf9.pitch;
  p9.dstPtr.xsize = M;
  p9.dstPtr.ysize = N;
  p9.extent.width = M * sizeof(float);
  p9.extent.height = N;
  p9.extent.depth = W;
  p9.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p9);

  p10.srcPtr.ptr = h_f10;
  p10.srcPtr.pitch = M * sizeof(float);
  p10.srcPtr.xsize = M;
  p10.srcPtr.ysize = N;
  p10.dstPtr.ptr = devPitchedf10.ptr;
  p10.dstPtr.pitch = devPitchedf10.pitch;
  p10.dstPtr.xsize = M;
  p10.dstPtr.ysize = N;
  p10.extent.width = M * sizeof(float);
  p10.extent.height = N;
  p10.extent.depth = W;
  p10.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p10);


  p11.srcPtr.ptr = h_f11;
  p11.srcPtr.pitch = M * sizeof(float);
  p11.srcPtr.xsize = M;
  p11.srcPtr.ysize = N;
  p11.dstPtr.ptr = devPitchedf11.ptr;
  p11.dstPtr.pitch = devPitchedf11.pitch;
  p11.dstPtr.xsize = M;
  p11.dstPtr.ysize = N;
  p11.extent.width = M * sizeof(float);
  p11.extent.height = N;
  p11.extent.depth = W;
  p11.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p11);


  p12.srcPtr.ptr = h_f12;
  p12.srcPtr.pitch = M * sizeof(float);
  p12.srcPtr.xsize = M;
  p12.srcPtr.ysize = N;
  p12.dstPtr.ptr = devPitchedf12.ptr;
  p12.dstPtr.pitch = devPitchedf12.pitch;
  p12.dstPtr.xsize = M;
  p12.dstPtr.ysize = N;
  p12.extent.width = M * sizeof(float);
  p12.extent.height = N;
  p12.extent.depth = W;
  p12.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p12);



  p13.srcPtr.ptr = h_f13;
  p13.srcPtr.pitch = M * sizeof(float);
  p13.srcPtr.xsize = M;
  p13.srcPtr.ysize = N;
  p13.dstPtr.ptr = devPitchedf13.ptr;
  p13.dstPtr.pitch = devPitchedf13.pitch;
  p13.dstPtr.xsize = M;
  p13.dstPtr.ysize = N;
  p13.extent.width = M * sizeof(float);
  p13.extent.height = N;
  p13.extent.depth = W;
  p13.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p13);


  p14.srcPtr.ptr = h_f14;
  p14.srcPtr.pitch = M * sizeof(float);
  p14.srcPtr.xsize = M;
  p14.srcPtr.ysize = N;
  p14.dstPtr.ptr = devPitchedf14.ptr;
  p14.dstPtr.pitch = devPitchedf14.pitch;
  p14.dstPtr.xsize = M;
  p14.dstPtr.ysize = N;
  p14.extent.width = M * sizeof(float);
  p14.extent.height = N;
  p14.extent.depth = W;
  p14.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p14);


  p15.srcPtr.ptr = h_f15;
  p15.srcPtr.pitch = M * sizeof(float);
  p15.srcPtr.xsize = M;
  p15.srcPtr.ysize = N;
  p15.dstPtr.ptr = devPitchedf15.ptr;
  p15.dstPtr.pitch = devPitchedf15.pitch;
  p15.dstPtr.xsize = M;
  p15.dstPtr.ysize = N;
  p15.extent.width = M * sizeof(float);
  p15.extent.height = N;
  p15.extent.depth = W;
  p15.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p15);

  p16.srcPtr.ptr = h_f16;
  p16.srcPtr.pitch = M * sizeof(float);
  p16.srcPtr.xsize = M;
  p16.srcPtr.ysize = N;
  p16.dstPtr.ptr = devPitchedf16.ptr;
  p16.dstPtr.pitch = devPitchedf16.pitch;
  p16.dstPtr.xsize = M;
  p16.dstPtr.ysize = N;
  p16.extent.width = M * sizeof(float);
  p16.extent.height = N;
  p16.extent.depth = W;
  p16.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p16);

  p17.srcPtr.ptr = h_f17;
  p17.srcPtr.pitch = M * sizeof(float);
  p17.srcPtr.xsize = M;
  p17.srcPtr.ysize = N;
  p17.dstPtr.ptr = devPitchedf17.ptr;
  p17.dstPtr.pitch = devPitchedf17.pitch;
  p17.dstPtr.xsize = M;
  p17.dstPtr.ysize = N;
  p17.extent.width = M * sizeof(float);
  p17.extent.height = N;
  p17.extent.depth = W;
  p17.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p17);

  p18.srcPtr.ptr = h_f18;
  p18.srcPtr.pitch = M * sizeof(float);
  p18.srcPtr.xsize = M;
  p18.srcPtr.ysize = N;
  p18.dstPtr.ptr = devPitchedf18.ptr;
  p18.dstPtr.pitch = devPitchedf18.pitch;
  p18.dstPtr.xsize = M;
  p18.dstPtr.ysize = N;
  p18.extent.width = M * sizeof(float);
  p18.extent.height = N;
  p18.extent.depth = W;
  p18.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p18);

  p0new.srcPtr.ptr = h_f0new;
  p0new.srcPtr.pitch = M * sizeof(float);
  p0new.srcPtr.xsize = M;
  p0new.srcPtr.ysize = N;
  p0new.dstPtr.ptr = devPitchedf0new.ptr;
  p0new.dstPtr.pitch = devPitchedf0new.pitch;
  p0new.dstPtr.xsize = M;
  p0new.dstPtr.ysize = N;
  p0new.extent.width = M * sizeof(float);
  p0new.extent.height = N;
  p0new.extent.depth = W;
  p0new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p0new);

  p1new.srcPtr.ptr = h_f1new;
  p1new.srcPtr.pitch = M * sizeof(float);
  p1new.srcPtr.xsize = M;
  p1new.srcPtr.ysize = N;
  p1new.dstPtr.ptr = devPitchedf1new.ptr;
  p1new.dstPtr.pitch = devPitchedf1new.pitch;
  p1new.dstPtr.xsize = M;
  p1new.dstPtr.ysize = N;
  p1new.extent.width = M * sizeof(float);
  p1new.extent.height = N;
  p1new.extent.depth = W;
  p1new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p1new);

  p2new.srcPtr.ptr = h_f2new;
  p2new.srcPtr.pitch = M * sizeof(float);
  p2new.srcPtr.xsize = M;
  p2new.srcPtr.ysize = N;
  p2new.dstPtr.ptr = devPitchedf2new.ptr;
  p2new.dstPtr.pitch = devPitchedf2new.pitch;
  p2new.dstPtr.xsize = M;
  p2new.dstPtr.ysize = N;
  p2new.extent.width = M * sizeof(float);
  p2new.extent.height = N;
  p2new.extent.depth = W;
  p2new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p2new);

  p3new.srcPtr.ptr = h_f3new;
  p3new.srcPtr.pitch = M * sizeof(float);
  p3new.srcPtr.xsize = M;
  p3new.srcPtr.ysize = N;
  p3new.dstPtr.ptr = devPitchedf3new.ptr;
  p3new.dstPtr.pitch = devPitchedf3new.pitch;
  p3new.dstPtr.xsize = M;
  p3new.dstPtr.ysize = N;
  p3new.extent.width = M * sizeof(float);
  p3new.extent.height = N;
  p3new.extent.depth = W;
  p3new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p3new);

  p4new.srcPtr.ptr = h_f4new;
  p4new.srcPtr.pitch = M * sizeof(float);
  p4new.srcPtr.xsize = M;
  p4new.srcPtr.ysize = N;
  p4new.dstPtr.ptr = devPitchedf4new.ptr;
  p4new.dstPtr.pitch = devPitchedf4new.pitch;
  p4new.dstPtr.xsize = M;
  p4new.dstPtr.ysize = N;
  p4new.extent.width = M * sizeof(float);
  p4new.extent.height = N;
  p4new.extent.depth = W;
  p4new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p4new);

  p5new.srcPtr.ptr = h_f5new;
  p5new.srcPtr.pitch = M * sizeof(float);
  p5new.srcPtr.xsize = M;
  p5new.srcPtr.ysize = N;
  p5new.dstPtr.ptr = devPitchedf5new.ptr;
  p5new.dstPtr.pitch = devPitchedf5new.pitch;
  p5new.dstPtr.xsize = M;
  p5new.dstPtr.ysize = N;
  p5new.extent.width = M * sizeof(float);
  p5new.extent.height = N;
  p5new.extent.depth = W;
  p5new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p5new);

  p6new.srcPtr.ptr = h_f6new;
  p6new.srcPtr.pitch = M * sizeof(float);
  p6new.srcPtr.xsize = M;
  p6new.srcPtr.ysize = N;
  p6new.dstPtr.ptr = devPitchedf6new.ptr;
  p6new.dstPtr.pitch = devPitchedf6new.pitch;
  p6new.dstPtr.xsize = M;
  p6new.dstPtr.ysize = N;
  p6new.extent.width = M * sizeof(float);
  p6new.extent.height = N;
  p6new.extent.depth = W;
  p6new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p6new);

  p7new.srcPtr.ptr = h_f7new;
  p7new.srcPtr.pitch = M * sizeof(float);
  p7new.srcPtr.xsize = M;
  p7new.srcPtr.ysize = N;
  p7new.dstPtr.ptr = devPitchedf7new.ptr;
  p7new.dstPtr.pitch = devPitchedf7new.pitch;
  p7new.dstPtr.xsize = M;
  p7new.dstPtr.ysize = N;
  p7new.extent.width = M * sizeof(float);
  p7new.extent.height = N;
  p7new.extent.depth = W;
  p7new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p7new);

  p8new.srcPtr.ptr = h_f8new;
  p8new.srcPtr.pitch = M * sizeof(float);
  p8new.srcPtr.xsize = M;
  p8new.srcPtr.ysize = N;
  p8new.dstPtr.ptr = devPitchedf8new.ptr;
  p8new.dstPtr.pitch = devPitchedf8new.pitch;
  p8new.dstPtr.xsize = M;
  p8new.dstPtr.ysize = N;
  p8new.extent.width = M * sizeof(float);
  p8new.extent.height = N;
  p8new.extent.depth = W;
  p8new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p8new);

  p9new.srcPtr.ptr = h_f9new;
  p9new.srcPtr.pitch = M * sizeof(float);
  p9new.srcPtr.xsize = M;
  p9new.srcPtr.ysize = N;
  p9new.dstPtr.ptr = devPitchedf9new.ptr;
  p9new.dstPtr.pitch = devPitchedf9new.pitch;
  p9new.dstPtr.xsize = M;
  p9new.dstPtr.ysize = N;
  p9new.extent.width = M * sizeof(float);
  p9new.extent.height = N;
  p9new.extent.depth = W;
  p9new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p9new);

  p10new.srcPtr.ptr = h_f10new;
  p10new.srcPtr.pitch = M * sizeof(float);
  p10new.srcPtr.xsize = M;
  p10new.srcPtr.ysize = N;
  p10new.dstPtr.ptr = devPitchedf10new.ptr;
  p10new.dstPtr.pitch = devPitchedf10new.pitch;
  p10new.dstPtr.xsize = M;
  p10new.dstPtr.ysize = N;
  p10new.extent.width = M * sizeof(float);
  p10new.extent.height = N;
  p10new.extent.depth = W;
  p10new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p10new);

  p11new.srcPtr.ptr = h_f11new;
  p11new.srcPtr.pitch = M * sizeof(float);
  p11new.srcPtr.xsize = M;
  p11new.srcPtr.ysize = N;
  p11new.dstPtr.ptr = devPitchedf11new.ptr;
  p11new.dstPtr.pitch = devPitchedf11new.pitch;
  p11new.dstPtr.xsize = M;
  p11new.dstPtr.ysize = N;
  p11new.extent.width = M * sizeof(float);
  p11new.extent.height = N;
  p11new.extent.depth = W;
  p11new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p11new);

  p12new.srcPtr.ptr = h_f12new;
  p12new.srcPtr.pitch = M * sizeof(float);
  p12new.srcPtr.xsize = M;
  p12new.srcPtr.ysize = N;
  p12new.dstPtr.ptr = devPitchedf12new.ptr;
  p12new.dstPtr.pitch = devPitchedf12new.pitch;
  p12new.dstPtr.xsize = M;
  p12new.dstPtr.ysize = N;
  p12new.extent.width = M * sizeof(float);
  p12new.extent.height = N;
  p12new.extent.depth = W;
  p12new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p12new);

  p13new.srcPtr.ptr = h_f13new;
  p13new.srcPtr.pitch = M * sizeof(float);
  p13new.srcPtr.xsize = M;
  p13new.srcPtr.ysize = N;
  p13new.dstPtr.ptr = devPitchedf13new.ptr;
  p13new.dstPtr.pitch = devPitchedf13new.pitch;
  p13new.dstPtr.xsize = M;
  p13new.dstPtr.ysize = N;
  p13new.extent.width = M * sizeof(float);
  p13new.extent.height = N;
  p13new.extent.depth = W;
  p13new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p13new);

  p14new.srcPtr.ptr = h_f14new;
  p14new.srcPtr.pitch = M * sizeof(float);
  p14new.srcPtr.xsize = M;
  p14new.srcPtr.ysize = N;
  p14new.dstPtr.ptr = devPitchedf14new.ptr;
  p14new.dstPtr.pitch = devPitchedf14new.pitch;
  p14new.dstPtr.xsize = M;
  p14new.dstPtr.ysize = N;
  p14new.extent.width = M * sizeof(float);
  p14new.extent.height = N;
  p14new.extent.depth = W;
  p14new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p14new);

  p15new.srcPtr.ptr = h_f15new;
  p15new.srcPtr.pitch = M * sizeof(float);
  p15new.srcPtr.xsize = M;
  p15new.srcPtr.ysize = N;
  p15new.dstPtr.ptr = devPitchedf15new.ptr;
  p15new.dstPtr.pitch = devPitchedf15new.pitch;
  p15new.dstPtr.xsize = M;
  p15new.dstPtr.ysize = N;
  p15new.extent.width = M * sizeof(float);
  p15new.extent.height = N;
  p15new.extent.depth = W;
  p15new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p15new);

  p16new.srcPtr.ptr = h_f16new;
  p16new.srcPtr.pitch = M * sizeof(float);
  p16new.srcPtr.xsize = M;
  p16new.srcPtr.ysize = N;
  p16new.dstPtr.ptr = devPitchedf16new.ptr;
  p16new.dstPtr.pitch = devPitchedf16new.pitch;
  p16new.dstPtr.xsize = M;
  p16new.dstPtr.ysize = N;
  p16new.extent.width = M * sizeof(float);
  p16new.extent.height = N;
  p16new.extent.depth = W;
  p16new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p16new);

  p17new.srcPtr.ptr = h_f17new;
  p17new.srcPtr.pitch = M * sizeof(float);
  p17new.srcPtr.xsize = M;
  p17new.srcPtr.ysize = N;
  p17new.dstPtr.ptr = devPitchedf17new.ptr;
  p17new.dstPtr.pitch = devPitchedf17new.pitch;
  p17new.dstPtr.xsize = M;
  p17new.dstPtr.ysize = N;
  p17new.extent.width = M * sizeof(float);
  p17new.extent.height = N;
  p17new.extent.depth = W;
  p17new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p17new);

  p18new.srcPtr.ptr = h_f18new;
  p18new.srcPtr.pitch = M * sizeof(float);
  p18new.srcPtr.xsize = M;
  p18new.srcPtr.ysize = N;
  p18new.dstPtr.ptr = devPitchedf18new.ptr;
  p18new.dstPtr.pitch = devPitchedf18new.pitch;
  p18new.dstPtr.xsize = M;
  p18new.dstPtr.ysize = N;
  p18new.extent.width = M * sizeof(float);
  p18new.extent.height = N;
  p18new.extent.depth = W;
  p18new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&p18new);

  q0.srcPtr.ptr = h_g0;
  q0.srcPtr.pitch = M * sizeof(float);
  q0.srcPtr.xsize = M;
  q0.srcPtr.ysize = N;
  q0.dstPtr.ptr = devPitchedg0.ptr;
  q0.dstPtr.pitch = devPitchedg0.pitch;
  q0.dstPtr.xsize = M;
  q0.dstPtr.ysize = N;
  q0.extent.width = M * sizeof(float);
  q0.extent.height = N;
  q0.extent.depth = W;
  q0.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q0);

  q1.srcPtr.ptr = h_g1;
  q1.srcPtr.pitch = M * sizeof(float);
  q1.srcPtr.xsize = M;
  q1.srcPtr.ysize = N;
  q1.dstPtr.ptr = devPitchedg1.ptr;
  q1.dstPtr.pitch = devPitchedg1.pitch;
  q1.dstPtr.xsize = M;
  q1.dstPtr.ysize = N;
  q1.extent.width = M * sizeof(float);
  q1.extent.height = N;
  q1.extent.depth = W;
  q1.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q1);

  q2.srcPtr.ptr = h_g2;
  q2.srcPtr.pitch = M * sizeof(float);
  q2.srcPtr.xsize = M;
  q2.srcPtr.ysize = N;
  q2.dstPtr.ptr = devPitchedg2.ptr;
  q2.dstPtr.pitch = devPitchedg2.pitch;
  q2.dstPtr.xsize = M;
  q2.dstPtr.ysize = N;
  q2.extent.width = M * sizeof(float);
  q2.extent.height = N;
  q2.extent.depth = W;
  q2.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q2);

  q3.srcPtr.ptr = h_g3;
  q3.srcPtr.pitch = M * sizeof(float);
  q3.srcPtr.xsize = M;
  q3.srcPtr.ysize = N;
  q3.dstPtr.ptr = devPitchedg3.ptr;
  q3.dstPtr.pitch = devPitchedg3.pitch;
  q3.dstPtr.xsize = M;
  q3.dstPtr.ysize = N;
  q3.extent.width = M * sizeof(float);
  q3.extent.height = N;
  q3.extent.depth = W;
  q3.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q3);

  q4.srcPtr.ptr = h_g4;
  q4.srcPtr.pitch = M * sizeof(float);
  q4.srcPtr.xsize = M;
  q4.srcPtr.ysize = N;
  q4.dstPtr.ptr = devPitchedg4.ptr;
  q4.dstPtr.pitch = devPitchedg4.pitch;
  q4.dstPtr.xsize = M;
  q4.dstPtr.ysize = N;
  q4.extent.width = M * sizeof(float);
  q4.extent.height = N;
  q4.extent.depth = W;
  q4.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q4);

  q5.srcPtr.ptr = h_g5;
  q5.srcPtr.pitch = M * sizeof(float);
  q5.srcPtr.xsize = M;
  q5.srcPtr.ysize = N;
  q5.dstPtr.ptr = devPitchedg5.ptr;
  q5.dstPtr.pitch = devPitchedg5.pitch;
  q5.dstPtr.xsize = M;
  q5.dstPtr.ysize = N;
  q5.extent.width = M * sizeof(float);
  q5.extent.height = N;
  q5.extent.depth = W;
  q5.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q5);

  q6.srcPtr.ptr = h_g6;
  q6.srcPtr.pitch = M * sizeof(float);
  q6.srcPtr.xsize = M;
  q6.srcPtr.ysize = N;
  q6.dstPtr.ptr = devPitchedg6.ptr;
  q6.dstPtr.pitch = devPitchedg6.pitch;
  q6.dstPtr.xsize = M;
  q6.dstPtr.ysize = N;
  q6.extent.width = M * sizeof(float);
  q6.extent.height = N;
  q6.extent.depth = W;
  q6.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q6);

  q7.srcPtr.ptr = h_g7;
  q7.srcPtr.pitch = M * sizeof(float);
  q7.srcPtr.xsize = M;
  q7.srcPtr.ysize = N;
  q7.dstPtr.ptr = devPitchedg7.ptr;
  q7.dstPtr.pitch = devPitchedg7.pitch;
  q7.dstPtr.xsize = M;
  q7.dstPtr.ysize = N;
  q7.extent.width = M * sizeof(float);
  q7.extent.height = N;
  q7.extent.depth = W;
  q7.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q7);

  q8.srcPtr.ptr = h_g8;
  q8.srcPtr.pitch = M * sizeof(float);
  q8.srcPtr.xsize = M;
  q8.srcPtr.ysize = N;
  q8.dstPtr.ptr = devPitchedg8.ptr;
  q8.dstPtr.pitch = devPitchedg8.pitch;
  q8.dstPtr.xsize = M;
  q8.dstPtr.ysize = N;
  q8.extent.width = M * sizeof(float);
  q8.extent.height = N;
  q8.extent.depth = W;
  q8.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q8);

  q9.srcPtr.ptr = h_g9;
  q9.srcPtr.pitch = M * sizeof(float);
  q9.srcPtr.xsize = M;
  q9.srcPtr.ysize = N;
  q9.dstPtr.ptr = devPitchedg9.ptr;
  q9.dstPtr.pitch = devPitchedg9.pitch;
  q9.dstPtr.xsize = M;
  q9.dstPtr.ysize = N;
  q9.extent.width = M * sizeof(float);
  q9.extent.height = N;
  q9.extent.depth = W;
  q9.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q9);

  q10.srcPtr.ptr = h_g10;
  q10.srcPtr.pitch = M * sizeof(float);
  q10.srcPtr.xsize = M;
  q10.srcPtr.ysize = N;
  q10.dstPtr.ptr = devPitchedg10.ptr;
  q10.dstPtr.pitch = devPitchedg10.pitch;
  q10.dstPtr.xsize = M;
  q10.dstPtr.ysize = N;
  q10.extent.width = M * sizeof(float);
  q10.extent.height = N;
  q10.extent.depth = W;
  q10.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q10);

  q11.srcPtr.ptr = h_g11;
  q11.srcPtr.pitch = M * sizeof(float);
  q11.srcPtr.xsize = M;
  q11.srcPtr.ysize = N;
  q11.dstPtr.ptr = devPitchedg11.ptr;
  q11.dstPtr.pitch = devPitchedg11.pitch;
  q11.dstPtr.xsize = M;
  q11.dstPtr.ysize = N;
  q11.extent.width = M * sizeof(float);
  q11.extent.height = N;
  q11.extent.depth = W;
  q11.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q11);

  q12.srcPtr.ptr = h_g12;
  q12.srcPtr.pitch = M * sizeof(float);
  q12.srcPtr.xsize = M;
  q12.srcPtr.ysize = N;
  q12.dstPtr.ptr = devPitchedg12.ptr;
  q12.dstPtr.pitch = devPitchedg12.pitch;
  q12.dstPtr.xsize = M;
  q12.dstPtr.ysize = N;
  q12.extent.width = M * sizeof(float);
  q12.extent.height = N;
  q12.extent.depth = W;
  q12.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q12);

  q13.srcPtr.ptr = h_g13;
  q13.srcPtr.pitch = M * sizeof(float);
  q13.srcPtr.xsize = M;
  q13.srcPtr.ysize = N;
  q13.dstPtr.ptr = devPitchedg13.ptr;
  q13.dstPtr.pitch = devPitchedg13.pitch;
  q13.dstPtr.xsize = M;
  q13.dstPtr.ysize = N;
  q13.extent.width = M * sizeof(float);
  q13.extent.height = N;
  q13.extent.depth = W;
  q13.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q13);

  q14.srcPtr.ptr = h_g14;
  q14.srcPtr.pitch = M * sizeof(float);
  q14.srcPtr.xsize = M;
  q14.srcPtr.ysize = N;
  q14.dstPtr.ptr = devPitchedg14.ptr;
  q14.dstPtr.pitch = devPitchedg14.pitch;
  q14.dstPtr.xsize = M;
  q14.dstPtr.ysize = N;
  q14.extent.width = M * sizeof(float);
  q14.extent.height = N;
  q14.extent.depth = W;
  q14.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q14);

  q15.srcPtr.ptr = h_g15;
  q15.srcPtr.pitch = M * sizeof(float);
  q15.srcPtr.xsize = M;
  q15.srcPtr.ysize = N;
  q15.dstPtr.ptr = devPitchedg15.ptr;
  q15.dstPtr.pitch = devPitchedg15.pitch;
  q15.dstPtr.xsize = M;
  q15.dstPtr.ysize = N;
  q15.extent.width = M * sizeof(float);
  q15.extent.height = N;
  q15.extent.depth = W;
  q15.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q15);

  q16.srcPtr.ptr = h_g16;
  q16.srcPtr.pitch = M * sizeof(float);
  q16.srcPtr.xsize = M;
  q16.srcPtr.ysize = N;
  q16.dstPtr.ptr = devPitchedg16.ptr;
  q16.dstPtr.pitch = devPitchedg16.pitch;
  q16.dstPtr.xsize = M;
  q16.dstPtr.ysize = N;
  q16.extent.width = M * sizeof(float);
  q16.extent.height = N;
  q16.extent.depth = W;
  q16.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q16);


  q17.srcPtr.ptr = h_g17;
  q17.srcPtr.pitch = M * sizeof(float);
  q17.srcPtr.xsize = M;
  q17.srcPtr.ysize = N;
  q17.dstPtr.ptr = devPitchedg17.ptr;
  q17.dstPtr.pitch = devPitchedg17.pitch;
  q17.dstPtr.xsize = M;
  q17.dstPtr.ysize = N;
  q17.extent.width = M * sizeof(float);
  q17.extent.height = N;
  q17.extent.depth = W;
  q17.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q17);

  q18.srcPtr.ptr = h_g18;
  q18.srcPtr.pitch = M * sizeof(float);
  q18.srcPtr.xsize = M;
  q18.srcPtr.ysize = N;
  q18.dstPtr.ptr = devPitchedg18.ptr;
  q18.dstPtr.pitch = devPitchedg18.pitch;
  q18.dstPtr.xsize = M;
  q18.dstPtr.ysize = N;
  q18.extent.width = M * sizeof(float);
  q18.extent.height = N;
  q18.extent.depth = W;
  q18.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q18);

  q0new.srcPtr.ptr = h_g0new;
  q0new.srcPtr.pitch = M * sizeof(float);
  q0new.srcPtr.xsize = M;
  q0new.srcPtr.ysize = N;
  q0new.dstPtr.ptr = devPitchedg0new.ptr;
  q0new.dstPtr.pitch = devPitchedg0new.pitch;
  q0new.dstPtr.xsize = M;
  q0new.dstPtr.ysize = N;
  q0new.extent.width = M * sizeof(float);
  q0new.extent.height = N;
  q0new.extent.depth = W;
  q0new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q0new);

  q1new.srcPtr.ptr = h_g1new;
  q1new.srcPtr.pitch = M * sizeof(float);
  q1new.srcPtr.xsize = M;
  q1new.srcPtr.ysize = N;
  q1new.dstPtr.ptr = devPitchedg1new.ptr;
  q1new.dstPtr.pitch = devPitchedg1new.pitch;
  q1new.dstPtr.xsize = M;
  q1new.dstPtr.ysize = N;
  q1new.extent.width = M * sizeof(float);
  q1new.extent.height = N;
  q1new.extent.depth = W;
  q1new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q1new);

  q2new.srcPtr.ptr = h_g2new;
  q2new.srcPtr.pitch = M * sizeof(float);
  q2new.srcPtr.xsize = M;
  q2new.srcPtr.ysize = N;
  q2new.dstPtr.ptr = devPitchedg2new.ptr;
  q2new.dstPtr.pitch = devPitchedg2new.pitch;
  q2new.dstPtr.xsize = M;
  q2new.dstPtr.ysize = N;
  q2new.extent.width = M * sizeof(float);
  q2new.extent.height = N;
  q2new.extent.depth = W;
  q2new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q2new);

  q3new.srcPtr.ptr = h_g3new;
  q3new.srcPtr.pitch = M * sizeof(float);
  q3new.srcPtr.xsize = M;
  q3new.srcPtr.ysize = N;
  q3new.dstPtr.ptr = devPitchedg3new.ptr;
  q3new.dstPtr.pitch = devPitchedg3new.pitch;
  q3new.dstPtr.xsize = M;
  q3new.dstPtr.ysize = N;
  q3new.extent.width = M * sizeof(float);
  q3new.extent.height = N;
  q3new.extent.depth = W;
  q3new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q3new);

  q4new.srcPtr.ptr = h_g4new;
  q4new.srcPtr.pitch = M * sizeof(float);
  q4new.srcPtr.xsize = M;
  q4new.srcPtr.ysize = N;
  q4new.dstPtr.ptr = devPitchedg4new.ptr;
  q4new.dstPtr.pitch = devPitchedg4new.pitch;
  q4new.dstPtr.xsize = M;
  q4new.dstPtr.ysize = N;
  q4new.extent.width = M * sizeof(float);
  q4new.extent.height = N;
  q4new.extent.depth = W;
  q4new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q4new);

  q5new.srcPtr.ptr = h_g5new;
  q5new.srcPtr.pitch = M * sizeof(float);
  q5new.srcPtr.xsize = M;
  q5new.srcPtr.ysize = N;
  q5new.dstPtr.ptr = devPitchedg5new.ptr;
  q5new.dstPtr.pitch = devPitchedg5new.pitch;
  q5new.dstPtr.xsize = M;
  q5new.dstPtr.ysize = N;
  q5new.extent.width = M * sizeof(float);
  q5new.extent.height = N;
  q5new.extent.depth = W;
  q5new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q5new);

  q6new.srcPtr.ptr = h_g6new;
  q6new.srcPtr.pitch = M * sizeof(float);
  q6new.srcPtr.xsize = M;
  q6new.srcPtr.ysize = N;
  q6new.dstPtr.ptr = devPitchedg6new.ptr;
  q6new.dstPtr.pitch = devPitchedg6new.pitch;
  q6new.dstPtr.xsize = M;
  q6new.dstPtr.ysize = N;
  q6new.extent.width = M * sizeof(float);
  q6new.extent.height = N;
  q6new.extent.depth = W;
  q6new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q6new);

  q7new.srcPtr.ptr = h_g7new;
  q7new.srcPtr.pitch = M * sizeof(float);
  q7new.srcPtr.xsize = M;
  q7new.srcPtr.ysize = N;
  q7new.dstPtr.ptr = devPitchedg7new.ptr;
  q7new.dstPtr.pitch = devPitchedg7new.pitch;
  q7new.dstPtr.xsize = M;
  q7new.dstPtr.ysize = N;
  q7new.extent.width = M * sizeof(float);
  q7new.extent.height = N;
  q7new.extent.depth = W;
  q7new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q7new);

  q8new.srcPtr.ptr = h_g8new;
  q8new.srcPtr.pitch = M * sizeof(float);
  q8new.srcPtr.xsize = M;
  q8new.srcPtr.ysize = N;
  q8new.dstPtr.ptr = devPitchedg8new.ptr;
  q8new.dstPtr.pitch = devPitchedg8new.pitch;
  q8new.dstPtr.xsize = M;
  q8new.dstPtr.ysize = N;
  q8new.extent.width = M * sizeof(float);
  q8new.extent.height = N;
  q8new.extent.depth = W;
  q8new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q8new);

  q9new.srcPtr.ptr = h_g9new;
  q9new.srcPtr.pitch = M * sizeof(float);
  q9new.srcPtr.xsize = M;
  q9new.srcPtr.ysize = N;
  q9new.dstPtr.ptr = devPitchedg9new.ptr;
  q9new.dstPtr.pitch = devPitchedg9new.pitch;
  q9new.dstPtr.xsize = M;
  q9new.dstPtr.ysize = N;
  q9new.extent.width = M * sizeof(float);
  q9new.extent.height = N;
  q9new.extent.depth = W;
  q9new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q9new);

  q10new.srcPtr.ptr = h_g10new;
  q10new.srcPtr.pitch = M * sizeof(float);
  q10new.srcPtr.xsize = M;
  q10new.srcPtr.ysize = N;
  q10new.dstPtr.ptr = devPitchedg10new.ptr;
  q10new.dstPtr.pitch = devPitchedg10new.pitch;
  q10new.dstPtr.xsize = M;
  q10new.dstPtr.ysize = N;
  q10new.extent.width = M * sizeof(float);
  q10new.extent.height = N;
  q10new.extent.depth = W;
  q10new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q10new);

  q11new.srcPtr.ptr = h_g11new;
  q11new.srcPtr.pitch = M * sizeof(float);
  q11new.srcPtr.xsize = M;
  q11new.srcPtr.ysize = N;
  q11new.dstPtr.ptr = devPitchedg11new.ptr;
  q11new.dstPtr.pitch = devPitchedg11new.pitch;
  q11new.dstPtr.xsize = M;
  q11new.dstPtr.ysize = N;
  q11new.extent.width = M * sizeof(float);
  q11new.extent.height = N;
  q11new.extent.depth = W;
  q11new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q11new);

  q12new.srcPtr.ptr = h_g12new;
  q12new.srcPtr.pitch = M * sizeof(float);
  q12new.srcPtr.xsize = M;
  q12new.srcPtr.ysize = N;
  q12new.dstPtr.ptr = devPitchedg12new.ptr;
  q12new.dstPtr.pitch = devPitchedg12new.pitch;
  q12new.dstPtr.xsize = M;
  q12new.dstPtr.ysize = N;
  q12new.extent.width = M * sizeof(float);
  q12new.extent.height = N;
  q12new.extent.depth = W;
  q12new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q12new);

  q13new.srcPtr.ptr = h_g13new;
  q13new.srcPtr.pitch = M * sizeof(float);
  q13new.srcPtr.xsize = M;
  q13new.srcPtr.ysize = N;
  q13new.dstPtr.ptr = devPitchedg13new.ptr;
  q13new.dstPtr.pitch = devPitchedg13new.pitch;
  q13new.dstPtr.xsize = M;
  q13new.dstPtr.ysize = N;
  q13new.extent.width = M * sizeof(float);
  q13new.extent.height = N;
  q13new.extent.depth = W;
  q13new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q13new);

  q14new.srcPtr.ptr = h_g14new;
  q14new.srcPtr.pitch = M * sizeof(float);
  q14new.srcPtr.xsize = M;
  q14new.srcPtr.ysize = N;
  q14new.dstPtr.ptr = devPitchedg14new.ptr;
  q14new.dstPtr.pitch = devPitchedg14new.pitch;
  q14new.dstPtr.xsize = M;
  q14new.dstPtr.ysize = N;
  q14new.extent.width = M * sizeof(float);
  q14new.extent.height = N;
  q14new.extent.depth = W;
  q14new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q14new);

  q15new.srcPtr.ptr = h_g15new;
  q15new.srcPtr.pitch = M * sizeof(float);
  q15new.srcPtr.xsize = M;
  q15new.srcPtr.ysize = N;
  q15new.dstPtr.ptr = devPitchedg15new.ptr;
  q15new.dstPtr.pitch = devPitchedg15new.pitch;
  q15new.dstPtr.xsize = M;
  q15new.dstPtr.ysize = N;
  q15new.extent.width = M * sizeof(float);
  q15new.extent.height = N;
  q15new.extent.depth = W;
  q15new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q15new);

  q16new.srcPtr.ptr = h_g16new;
  q16new.srcPtr.pitch = M * sizeof(float);
  q16new.srcPtr.xsize = M;
  q16new.srcPtr.ysize = N;
  q16new.dstPtr.ptr = devPitchedg16new.ptr;
  q16new.dstPtr.pitch = devPitchedg16new.pitch;
  q16new.dstPtr.xsize = M;
  q16new.dstPtr.ysize = N;
  q16new.extent.width = M * sizeof(float);
  q16new.extent.height = N;
  q16new.extent.depth = W;
  q16new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q16new);

  q17new.srcPtr.ptr = h_g17new;
  q17new.srcPtr.pitch = M * sizeof(float);
  q17new.srcPtr.xsize = M;
  q17new.srcPtr.ysize = N;
  q17new.dstPtr.ptr = devPitchedg17new.ptr;
  q17new.dstPtr.pitch = devPitchedg17new.pitch;
  q17new.dstPtr.xsize = M;
  q17new.dstPtr.ysize = N;
  q17new.extent.width = M * sizeof(float);
  q17new.extent.height = N;
  q17new.extent.depth = W;
  q17new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q17new);

  q18new.srcPtr.ptr = h_g18new;
  q18new.srcPtr.pitch = M * sizeof(float);
  q18new.srcPtr.xsize = M;
  q18new.srcPtr.ysize = N;
  q18new.dstPtr.ptr = devPitchedg18new.ptr;
  q18new.dstPtr.pitch = devPitchedg18new.pitch;
  q18new.dstPtr.xsize = M;
  q18new.dstPtr.ysize = N;
  q18new.extent.width = M * sizeof(float);
  q18new.extent.height = N;
  q18new.extent.depth = W;
  q18new.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&q18new);
}
void LatticeBoltzmann::Collision(void){
  dim3 GridSize_(Mx,My,Mz);
  dim3 BlockSize_(BLOCKSIZE_x,BLOCKSIZE_y,BLOCKSIZE_z);
  d_collition<<<GridSize_,BlockSize_>>>(devPitchedf0,devPitchedf0new,devPitchedg0,devPitchedg0new,
                                        devPitchedf1,devPitchedf1new,devPitchedg1,devPitchedg1new,
                                        devPitchedf2,devPitchedf2new,devPitchedg2,devPitchedg2new,
                                        devPitchedf3,devPitchedf3new,devPitchedg3,devPitchedg3new,
                                        devPitchedf4,devPitchedf4new,devPitchedg4,devPitchedg4new,
                                        devPitchedf5,devPitchedf5new,devPitchedg5,devPitchedg5new,
                                        devPitchedf6,devPitchedf6new,devPitchedg6,devPitchedg6new,
                                        devPitchedf7,devPitchedf7new,devPitchedg7,devPitchedg7new,
                                        devPitchedf8,devPitchedf8new,devPitchedg8,devPitchedg8new,
                                        devPitchedf9,devPitchedf9new,devPitchedg9,devPitchedg9new,
                                        devPitchedf10,devPitchedf10new,devPitchedg10,devPitchedg10new,
                                        devPitchedf11,devPitchedf11new,devPitchedg11,devPitchedg11new,
                                        devPitchedf12,devPitchedf12new,devPitchedg12,devPitchedg12new,
                                        devPitchedf13,devPitchedf13new,devPitchedg13,devPitchedg13new,
                                        devPitchedf14,devPitchedf14new,devPitchedg14,devPitchedg14new,
                                        devPitchedf15,devPitchedf15new,devPitchedg15,devPitchedg15new,
                                        devPitchedf16,devPitchedf16new,devPitchedg16,devPitchedg16new,
                                        devPitchedf17,devPitchedf17new,devPitchedg17,devPitchedg17new,
                                        devPitchedf18,devPitchedf18new,devPitchedg18,devPitchedg18new);
  hipDeviceSynchronize();
}
void LatticeBoltzmann::Advection(void){
  dim3 GridSize(Mx,My,Mz);
  dim3 BlockSize(BLOCKSIZE_x,BLOCKSIZE_y,BLOCKSIZE_z);
  
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf0,devPitchedf0new,0);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg0,devPitchedg0new,0);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf1,devPitchedf1new,1);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg1,devPitchedg1new,1);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf2,devPitchedf2new,2);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg2,devPitchedg2new,2);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf3,devPitchedf3new,3);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg3,devPitchedg3new,3);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf4,devPitchedf4new,4);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg4,devPitchedg4new,4);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf5,devPitchedf5new,5);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg5,devPitchedg5new,5);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf6,devPitchedf6new,6);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg6,devPitchedg6new,6);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf7,devPitchedf7new,7);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg7,devPitchedg7new,7);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf8,devPitchedf8new,8);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg8,devPitchedg8new,8);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf9,devPitchedf9new,9);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg9,devPitchedg9new,9);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf10,devPitchedf10new,10);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg10,devPitchedg10new,10);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf11,devPitchedf11new,11);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg11,devPitchedg11new,11);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf12,devPitchedf12new,12);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg12,devPitchedg12new,12);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf13,devPitchedf13new,13);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg13,devPitchedg13new,13);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf14,devPitchedf14new,14);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg14,devPitchedg14new,14);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf15,devPitchedf15new,15);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg15,devPitchedg15new,15);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf16,devPitchedf16new,16);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg16,devPitchedg16new,16);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf17,devPitchedf17new,17);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg17,devPitchedg17new,17);
  op_indv_advection<<<GridSize,BlockSize>>>(devPitchedf18,devPitchedf18new,18);   op_indv_advection<<<GridSize,BlockSize>>>(devPitchedg18,devPitchedg18new,18);
}

void LatticeBoltzmann::Show(void)
{
  //Devolver al Host
  p0.srcPtr.ptr = devPitchedf0.ptr;
  p0.srcPtr.pitch = devPitchedf0.pitch;
  p0.dstPtr.ptr = h_f0;
  p0.dstPtr.pitch = M * sizeof(float); 
  p0.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p0);

  p1.srcPtr.ptr = devPitchedf1.ptr;
  p1.srcPtr.pitch = devPitchedf1.pitch;
  p1.dstPtr.ptr = h_f1;
  p1.dstPtr.pitch = M * sizeof(float); 
  p1.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p1);

  p2.srcPtr.ptr = devPitchedf2.ptr;
  p2.srcPtr.pitch = devPitchedf2.pitch;
  p2.dstPtr.ptr = h_f2;
  p2.dstPtr.pitch = M * sizeof(float); 
  p2.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p2);

  p3.srcPtr.ptr = devPitchedf3.ptr;
  p3.srcPtr.pitch = devPitchedf3.pitch;
  p3.dstPtr.ptr = h_f3;
  p3.dstPtr.pitch = M * sizeof(float); 
  p3.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p3);

  p4.srcPtr.ptr = devPitchedf4.ptr;
  p4.srcPtr.pitch = devPitchedf4.pitch;
  p4.dstPtr.ptr = h_f4;
  p4.dstPtr.pitch = M * sizeof(float); 
  p4.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p4);

  p5.srcPtr.ptr = devPitchedf5.ptr;
  p5.srcPtr.pitch = devPitchedf5.pitch;
  p5.dstPtr.ptr = h_f5;
  p5.dstPtr.pitch = M * sizeof(float); 
  p5.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p5);

  p6.srcPtr.ptr = devPitchedf6.ptr;
  p6.srcPtr.pitch = devPitchedf6.pitch;
  p6.dstPtr.ptr = h_f6;
  p6.dstPtr.pitch = M * sizeof(float); 
  p6.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p6);

  p7.srcPtr.ptr = devPitchedf7.ptr;
  p7.srcPtr.pitch = devPitchedf7.pitch;
  p7.dstPtr.ptr = h_f7;
  p7.dstPtr.pitch = M * sizeof(float); 
  p7.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p7);

  p8.srcPtr.ptr = devPitchedf8.ptr;
  p8.srcPtr.pitch = devPitchedf8.pitch;
  p8.dstPtr.ptr = h_f8;
  p8.dstPtr.pitch = M * sizeof(float); 
  p8.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p8);

  p9.srcPtr.ptr = devPitchedf9.ptr;
  p9.srcPtr.pitch = devPitchedf9.pitch;
  p9.dstPtr.ptr = h_f9;
  p9.dstPtr.pitch = M * sizeof(float); 
  p9.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p9);

  p10.srcPtr.ptr = devPitchedf10.ptr;
  p10.srcPtr.pitch = devPitchedf10.pitch;
  p10.dstPtr.ptr = h_f10;
  p10.dstPtr.pitch = M * sizeof(float); 
  p10.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p10);

  p11.srcPtr.ptr = devPitchedf11.ptr;
  p11.srcPtr.pitch = devPitchedf11.pitch;
  p11.dstPtr.ptr = h_f11;
  p11.dstPtr.pitch = M * sizeof(float); 
  p11.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p11);

  p12.srcPtr.ptr = devPitchedf12.ptr;
  p12.srcPtr.pitch = devPitchedf12.pitch;
  p12.dstPtr.ptr = h_f12;
  p12.dstPtr.pitch = M * sizeof(float); 
  p12.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p12);

  p13.srcPtr.ptr = devPitchedf13.ptr;
  p13.srcPtr.pitch = devPitchedf13.pitch;
  p13.dstPtr.ptr = h_f13;
  p13.dstPtr.pitch = M * sizeof(float); 
  p13.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p13);

  p14.srcPtr.ptr = devPitchedf14.ptr;
  p14.srcPtr.pitch = devPitchedf14.pitch;
  p14.dstPtr.ptr = h_f14;
  p14.dstPtr.pitch = M * sizeof(float); 
  p14.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p14);


  p15.srcPtr.ptr = devPitchedf15.ptr;
  p15.srcPtr.pitch = devPitchedf15.pitch;
  p15.dstPtr.ptr = h_f15;
  p15.dstPtr.pitch = M * sizeof(float); 
  p15.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p15);

  p16.srcPtr.ptr = devPitchedf16.ptr;
  p16.srcPtr.pitch = devPitchedf16.pitch;
  p16.dstPtr.ptr = h_f16;
  p16.dstPtr.pitch = M * sizeof(float); 
  p16.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p16);

  p17.srcPtr.ptr = devPitchedf17.ptr;
  p17.srcPtr.pitch = devPitchedf17.pitch;
  p17.dstPtr.ptr = h_f17;
  p17.dstPtr.pitch = M * sizeof(float); 
  p17.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p17);


  p18.srcPtr.ptr = devPitchedf18.ptr;
  p18.srcPtr.pitch = devPitchedf18.pitch;
  p18.dstPtr.ptr = h_f18;
  p18.dstPtr.pitch = M * sizeof(float); 
  p18.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p18);

  q0.srcPtr.ptr = devPitchedg0.ptr;
  q0.srcPtr.pitch = devPitchedg0.pitch;
  q0.dstPtr.ptr = h_g0;
  q0.dstPtr.pitch = M * sizeof(float); 
  q0.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q0);

  q1.srcPtr.ptr = devPitchedg1.ptr;
  q1.srcPtr.pitch = devPitchedg1.pitch;
  q1.dstPtr.ptr = h_g1;
  q1.dstPtr.pitch = M * sizeof(float); 
  q1.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q1);

  q2.srcPtr.ptr = devPitchedg2.ptr;
  q2.srcPtr.pitch = devPitchedg2.pitch;
  q2.dstPtr.ptr = h_g2;
  q2.dstPtr.pitch = M * sizeof(float); 
  q2.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q2);

  q3.srcPtr.ptr = devPitchedg3.ptr;
  q3.srcPtr.pitch = devPitchedg3.pitch;
  q3.dstPtr.ptr = h_g3;
  q3.dstPtr.pitch = M * sizeof(float); 
  q3.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q3);

  q4.srcPtr.ptr = devPitchedg4.ptr;
  q4.srcPtr.pitch = devPitchedg4.pitch;
  q4.dstPtr.ptr = h_g4;
  q4.dstPtr.pitch = M * sizeof(float); 
  q4.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q4);

  q5.srcPtr.ptr = devPitchedg5.ptr;
  q5.srcPtr.pitch = devPitchedg5.pitch;
  q5.dstPtr.ptr = h_g5;
  q5.dstPtr.pitch = M * sizeof(float); 
  q5.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q5);

  q6.srcPtr.ptr = devPitchedg6.ptr;
  q6.srcPtr.pitch = devPitchedg6.pitch;
  q6.dstPtr.ptr = h_g6;
  q6.dstPtr.pitch = M * sizeof(float); 
  q6.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q6);

  q7.srcPtr.ptr = devPitchedg7.ptr;
  q7.srcPtr.pitch = devPitchedg7.pitch;
  q7.dstPtr.ptr = h_g7;
  q7.dstPtr.pitch = M * sizeof(float); 
  q7.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q7);

  q8.srcPtr.ptr = devPitchedg8.ptr;
  q8.srcPtr.pitch = devPitchedg8.pitch;
  q8.dstPtr.ptr = h_g8;
  q8.dstPtr.pitch = M * sizeof(float); 
  q8.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q8);

  q9.srcPtr.ptr = devPitchedg9.ptr;
  q9.srcPtr.pitch = devPitchedg9.pitch;
  q9.dstPtr.ptr = h_g9;
  q9.dstPtr.pitch = M * sizeof(float); 
  q9.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q9);

  q10.srcPtr.ptr = devPitchedg10.ptr;
  q10.srcPtr.pitch = devPitchedg10.pitch;
  q10.dstPtr.ptr = h_g10;
  q10.dstPtr.pitch = M * sizeof(float); 
  q10.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q10);

  q11.srcPtr.ptr = devPitchedg11.ptr;
  q11.srcPtr.pitch = devPitchedg11.pitch;
  q11.dstPtr.ptr = h_g11;
  q11.dstPtr.pitch = M * sizeof(float); 
  q11.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q11);

  q12.srcPtr.ptr = devPitchedg12.ptr;
  q12.srcPtr.pitch = devPitchedg12.pitch;
  q12.dstPtr.ptr = h_g12;
  q12.dstPtr.pitch = M * sizeof(float); 
  q12.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q12);

  q13.srcPtr.ptr = devPitchedg13.ptr;
  q13.srcPtr.pitch = devPitchedg13.pitch;
  q13.dstPtr.ptr = h_g13;
  q13.dstPtr.pitch = M * sizeof(float); 
  q13.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q13);

  q14.srcPtr.ptr = devPitchedg14.ptr;
  q14.srcPtr.pitch = devPitchedg14.pitch;
  q14.dstPtr.ptr = h_g14;
  q14.dstPtr.pitch = M * sizeof(float); 
  q14.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q14);


  q15.srcPtr.ptr = devPitchedg15.ptr;
  q15.srcPtr.pitch = devPitchedg15.pitch;
  q15.dstPtr.ptr = h_g15;
  q15.dstPtr.pitch = M * sizeof(float); 
  q15.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q15);

  q16.srcPtr.ptr = devPitchedg16.ptr;
  q16.srcPtr.pitch = devPitchedg16.pitch;
  q16.dstPtr.ptr = h_g16;
  q16.dstPtr.pitch = M * sizeof(float); 
  q16.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q16);

  q17.srcPtr.ptr = devPitchedg17.ptr;
  q17.srcPtr.pitch = devPitchedg17.pitch;
  q17.dstPtr.ptr = h_g17;
  q17.dstPtr.pitch = M * sizeof(float); 
  q17.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q17);


  q18.srcPtr.ptr = devPitchedg18.ptr;
  q18.srcPtr.pitch = devPitchedg18.pitch;
  q18.dstPtr.ptr = h_g18;
  q18.dstPtr.pitch = M * sizeof(float); 
  q18.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q18);

  //Devolver al Host
  p0new.srcPtr.ptr = devPitchedf0new.ptr;
  p0new.srcPtr.pitch = devPitchedf0new.pitch;
  p0new.dstPtr.ptr = h_f0new;
  p0new.dstPtr.pitch = M * sizeof(float); 
  p0new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p0new);

  p1new.srcPtr.ptr = devPitchedf1new.ptr;
  p1new.srcPtr.pitch = devPitchedf1new.pitch;
  p1new.dstPtr.ptr = h_f1new;
  p1new.dstPtr.pitch = M * sizeof(float); 
  p1new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p1new);

  p2new.srcPtr.ptr = devPitchedf2new.ptr;
  p2new.srcPtr.pitch = devPitchedf2new.pitch;
  p2new.dstPtr.ptr = h_f2new;
  p2new.dstPtr.pitch = M * sizeof(float); 
  p2new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p2new);

  p3new.srcPtr.ptr = devPitchedf3new.ptr;
  p3new.srcPtr.pitch = devPitchedf3new.pitch;
  p3new.dstPtr.ptr = h_f3new;
  p3new.dstPtr.pitch = M * sizeof(float); 
  p3new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p3new);

  p4new.srcPtr.ptr = devPitchedf4new.ptr;
  p4new.srcPtr.pitch = devPitchedf4new.pitch;
  p4new.dstPtr.ptr = h_f4new;
  p4new.dstPtr.pitch = M * sizeof(float); 
  p4new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p4new);

  p5new.srcPtr.ptr = devPitchedf5new.ptr;
  p5new.srcPtr.pitch = devPitchedf5new.pitch;
  p5new.dstPtr.ptr = h_f5new;
  p5new.dstPtr.pitch = M * sizeof(float); 
  p5new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p5new);

  p6new.srcPtr.ptr = devPitchedf6new.ptr;
  p6new.srcPtr.pitch = devPitchedf6new.pitch;
  p6new.dstPtr.ptr = h_f6new;
  p6new.dstPtr.pitch = M * sizeof(float); 
  p6new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p6new);

  p7new.srcPtr.ptr = devPitchedf7new.ptr;
  p7new.srcPtr.pitch = devPitchedf7new.pitch;
  p7new.dstPtr.ptr = h_f7new;
  p7new.dstPtr.pitch = M * sizeof(float); 
  p7new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p7new);

  p8new.srcPtr.ptr = devPitchedf8new.ptr;
  p8new.srcPtr.pitch = devPitchedf8new.pitch;
  p8new.dstPtr.ptr = h_f8new;
  p8new.dstPtr.pitch = M * sizeof(float); 
  p8new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p8new);

  p9new.srcPtr.ptr = devPitchedf9new.ptr;
  p9new.srcPtr.pitch = devPitchedf9new.pitch;
  p9new.dstPtr.ptr = h_f9new;
  p9new.dstPtr.pitch = M * sizeof(float); 
  p9new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p9new);

  p10new.srcPtr.ptr = devPitchedf10new.ptr;
  p10new.srcPtr.pitch = devPitchedf10new.pitch;
  p10new.dstPtr.ptr = h_f10new;
  p10new.dstPtr.pitch = M * sizeof(float); 
  p10new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p10new);

  p11new.srcPtr.ptr = devPitchedf11new.ptr;
  p11new.srcPtr.pitch = devPitchedf11new.pitch;
  p11new.dstPtr.ptr = h_f11new;
  p11new.dstPtr.pitch = M * sizeof(float); 
  p11new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p11new);

  p12new.srcPtr.ptr = devPitchedf12new.ptr;
  p12new.srcPtr.pitch = devPitchedf12new.pitch;
  p12new.dstPtr.ptr = h_f12new;
  p12new.dstPtr.pitch = M * sizeof(float); 
  p12new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p12new);

  p13new.srcPtr.ptr = devPitchedf13new.ptr;
  p13new.srcPtr.pitch = devPitchedf13new.pitch;
  p13new.dstPtr.ptr = h_f13new;
  p13new.dstPtr.pitch = M * sizeof(float); 
  p13new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p13new);

  p14new.srcPtr.ptr = devPitchedf14new.ptr;
  p14new.srcPtr.pitch = devPitchedf14new.pitch;
  p14new.dstPtr.ptr = h_f14new;
  p14new.dstPtr.pitch = M * sizeof(float); 
  p14new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p14new);


  p15new.srcPtr.ptr = devPitchedf15new.ptr;
  p15new.srcPtr.pitch = devPitchedf15new.pitch;
  p15new.dstPtr.ptr = h_f15new;
  p15new.dstPtr.pitch = M * sizeof(float); 
  p15new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p15new);

  p16new.srcPtr.ptr = devPitchedf16new.ptr;
  p16new.srcPtr.pitch = devPitchedf16new.pitch;
  p16new.dstPtr.ptr = h_f16new;
  p16new.dstPtr.pitch = M * sizeof(float); 
  p16new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p16new);

  p17new.srcPtr.ptr = devPitchedf17new.ptr;
  p17new.srcPtr.pitch = devPitchedf17new.pitch;
  p17new.dstPtr.ptr = h_f17new;
  p17new.dstPtr.pitch = M * sizeof(float); 
  p17new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p17new);


  p18new.srcPtr.ptr = devPitchedf18new.ptr;
  p18new.srcPtr.pitch = devPitchedf18new.pitch;
  p18new.dstPtr.ptr = h_f18new;
  p18new.dstPtr.pitch = M * sizeof(float); 
  p18new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&p18new);

  q0new.srcPtr.ptr = devPitchedg0new.ptr;
  q0new.srcPtr.pitch = devPitchedg0new.pitch;
  q0new.dstPtr.ptr = h_g0new;
  q0new.dstPtr.pitch = M * sizeof(float); 
  q0new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q0new);

  q1new.srcPtr.ptr = devPitchedg1new.ptr;
  q1new.srcPtr.pitch = devPitchedg1new.pitch;
  q1new.dstPtr.ptr = h_g1new;
  q1new.dstPtr.pitch = M * sizeof(float); 
  q1new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q1new);

  q2new.srcPtr.ptr = devPitchedg2new.ptr;
  q2new.srcPtr.pitch = devPitchedg2new.pitch;
  q2new.dstPtr.ptr = h_g2new;
  q2new.dstPtr.pitch = M * sizeof(float); 
  q2new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q2new);

  q3new.srcPtr.ptr = devPitchedg3new.ptr;
  q3new.srcPtr.pitch = devPitchedg3new.pitch;
  q3new.dstPtr.ptr = h_g3new;
  q3new.dstPtr.pitch = M * sizeof(float); 
  q3new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q3new);

  q4new.srcPtr.ptr = devPitchedg4new.ptr;
  q4new.srcPtr.pitch = devPitchedg4new.pitch;
  q4new.dstPtr.ptr = h_g4new;
  q4new.dstPtr.pitch = M * sizeof(float); 
  q4new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q4new);

  q5new.srcPtr.ptr = devPitchedg5new.ptr;
  q5new.srcPtr.pitch = devPitchedg5new.pitch;
  q5new.dstPtr.ptr = h_g5new;
  q5new.dstPtr.pitch = M * sizeof(float); 
  q5new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q5new);

  q6new.srcPtr.ptr = devPitchedg6new.ptr;
  q6new.srcPtr.pitch = devPitchedg6new.pitch;
  q6new.dstPtr.ptr = h_g6new;
  q6new.dstPtr.pitch = M * sizeof(float); 
  q6new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q6new);

  q7new.srcPtr.ptr = devPitchedg7new.ptr;
  q7new.srcPtr.pitch = devPitchedg7new.pitch;
  q7new.dstPtr.ptr = h_g7new;
  q7new.dstPtr.pitch = M * sizeof(float); 
  q7new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q7new);

  q8new.srcPtr.ptr = devPitchedg8new.ptr;
  q8new.srcPtr.pitch = devPitchedg8new.pitch;
  q8new.dstPtr.ptr = h_g8new;
  q8new.dstPtr.pitch = M * sizeof(float); 
  q8new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q8new);

  q9new.srcPtr.ptr = devPitchedg9new.ptr;
  q9new.srcPtr.pitch = devPitchedg9new.pitch;
  q9new.dstPtr.ptr = h_g9new;
  q9new.dstPtr.pitch = M * sizeof(float); 
  q9new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q9new);

  q10new.srcPtr.ptr = devPitchedg10new.ptr;
  q10new.srcPtr.pitch = devPitchedg10new.pitch;
  q10new.dstPtr.ptr = h_g10new;
  q10new.dstPtr.pitch = M * sizeof(float); 
  q10new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q10new);

  q11new.srcPtr.ptr = devPitchedg11new.ptr;
  q11new.srcPtr.pitch = devPitchedg11new.pitch;
  q11new.dstPtr.ptr = h_g11new;
  q11new.dstPtr.pitch = M * sizeof(float); 
  q11new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q11new);

  q12new.srcPtr.ptr = devPitchedg12new.ptr;
  q12new.srcPtr.pitch = devPitchedg12new.pitch;
  q12new.dstPtr.ptr = h_g12new;
  q12new.dstPtr.pitch = M * sizeof(float); 
  q12new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q12new);

  q13new.srcPtr.ptr = devPitchedg13new.ptr;
  q13new.srcPtr.pitch = devPitchedg13new.pitch;
  q13new.dstPtr.ptr = h_g13new;
  q13new.dstPtr.pitch = M * sizeof(float); 
  q13new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q13new);

  q14new.srcPtr.ptr = devPitchedg14new.ptr;
  q14new.srcPtr.pitch = devPitchedg14new.pitch;
  q14new.dstPtr.ptr = h_g14new;
  q14new.dstPtr.pitch = M * sizeof(float); 
  q14new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q14new);


  q15new.srcPtr.ptr = devPitchedg15new.ptr;
  q15new.srcPtr.pitch = devPitchedg15new.pitch;
  q15new.dstPtr.ptr = h_g15new;
  q15new.dstPtr.pitch = M * sizeof(float); 
  q15new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q15new);

  q16new.srcPtr.ptr = devPitchedg16new.ptr;
  q16new.srcPtr.pitch = devPitchedg16new.pitch;
  q16new.dstPtr.ptr = h_g16new;
  q16new.dstPtr.pitch = M * sizeof(float); 
  q16new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q16new);

  q17new.srcPtr.ptr = devPitchedg17new.ptr;
  q17new.srcPtr.pitch = devPitchedg17new.pitch;
  q17new.dstPtr.ptr = h_g17new;
  q17new.dstPtr.pitch = M * sizeof(float); 
  q17new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q17new);


  q18new.srcPtr.ptr = devPitchedg18new.ptr;
  q18new.srcPtr.pitch = devPitchedg18new.pitch;
  q18new.dstPtr.ptr = h_g18new;
  q18new.dstPtr.pitch = M * sizeof(float); 
  q18new.kind = hipMemcpyDeviceToHost;
  hipMemcpy3D(&q18new);
/*
    for (int w=0; w<W; w++) 
      for (int j=0; j<N; j++) 
        for (int i=0; i<M; i++){
         cout << h_g10[w][j][i] << " ";
         if(i == M -1){
           cout << endl;
         }
        }
    cout << endl;
*/
}


float LatticeBoltzmann::h_Ux(int ix,int iy,int iz){
  float sum1=0, sum2=0;
  
  sum2 = h_g0[iz][iy][ix]*h_Vx[0]+h_g1[iz][iy][ix]*h_Vx[1]+h_g2[iz][iy][ix]*h_Vx[2]+h_g3[iz][iy][ix]*h_Vx[3]+h_g4[iz][iy][ix]*h_Vx[4]+h_g5[iz][iy][ix]*h_Vx[5]+h_g6[iz][iy][ix]*h_Vx[6]+h_g7[iz][iy][ix]*h_Vx[7]+h_g8[iz][iy][ix]*h_Vx[8]+h_g9[iz][iy][ix]*h_Vx[9]+h_g10[iz][iy][ix]*h_Vx[10]+h_g11[iz][iy][ix]*h_Vx[11]+h_g12[iz][iy][ix]*h_Vx[12]+h_g13[iz][iy][ix]*h_Vx[13]+h_g14[iz][iy][ix]*h_Vx[14]+h_g15[iz][iy][ix]*h_Vx[15]+h_g16[iz][iy][ix]*h_Vx[16]+h_g17[iz][iy][ix]*h_Vx[17]+h_g18[iz][iy][ix]*h_Vx[18];
  
  sum1 = h_g0[iz][iy][ix]+h_g1[iz][iy][ix]+h_g2[iz][iy][ix]+h_g3[iz][iy][ix]+h_g4[iz][iy][ix]+h_g5[iz][iy][ix]+h_g6[iz][iy][ix]+h_g7[iz][iy][ix]+h_g8[iz][iy][ix]+h_g9[iz][iy][ix]+h_g10[iz][iy][ix]+h_g11[iz][iy][ix]+h_g12[iz][iy][ix]+h_g13[iz][iy][ix]+h_g14[iz][iy][ix]+h_g15[iz][iy][ix]+h_g16[iz][iy][ix]+h_g17[iz][iy][ix]+h_g18[iz][iy][ix];
  
  return 3.*sum2/(3.*sum1 + 3.*h_P(ix,iy,iz));
}
float LatticeBoltzmann::h_Uy(int ix,int iy,int iz){
  float sum1=0, sum2=0;
  
  sum2 = h_g0[iz][iy][ix]*h_Vy[0]+h_g1[iz][iy][ix]*h_Vy[1]+h_g2[iz][iy][ix]*h_Vy[2]+h_g3[iz][iy][ix]*h_Vy[3]+h_g4[iz][iy][ix]*h_Vy[4]+h_g5[iz][iy][ix]*h_Vy[5]+h_g6[iz][iy][ix]*h_Vy[6]+h_g7[iz][iy][ix]*h_Vy[7]+h_g8[iz][iy][ix]*h_Vy[8]+h_g9[iz][iy][ix]*h_Vy[9]+h_g10[iz][iy][ix]*h_Vy[10]+h_g11[iz][iy][ix]*h_Vy[11]+h_g12[iz][iy][ix]*h_Vy[12]+h_g13[iz][iy][ix]*h_Vy[13]+h_g14[iz][iy][ix]*h_Vy[14]+h_g15[iz][iy][ix]*h_Vy[15]+h_g16[iz][iy][ix]*h_Vy[16]+h_g17[iz][iy][ix]*h_Vy[17]+h_g18[iz][iy][ix]*h_Vy[18];
  
  sum1 = h_g0[iz][iy][ix]+h_g1[iz][iy][ix]+h_g2[iz][iy][ix]+h_g3[iz][iy][ix]+h_g4[iz][iy][ix]+h_g5[iz][iy][ix]+h_g6[iz][iy][ix]+h_g7[iz][iy][ix]+h_g8[iz][iy][ix]+h_g9[iz][iy][ix]+h_g10[iz][iy][ix]+h_g11[iz][iy][ix]+h_g12[iz][iy][ix]+h_g13[iz][iy][ix]+h_g14[iz][iy][ix]+h_g15[iz][iy][ix]+h_g16[iz][iy][ix]+h_g17[iz][iy][ix]+h_g18[iz][iy][ix];
  
  return 3.*sum2/(3.*sum1 + 3.*h_P(ix,iy,iz));
}
float LatticeBoltzmann::h_Uz(int ix,int iy,int iz){
  float sum1=0, sum2=0;
  
  sum2 = h_g0[iz][iy][ix]*h_Vz[0]+h_g1[iz][iy][ix]*h_Vz[1]+h_g2[iz][iy][ix]*h_Vz[2]+h_g3[iz][iy][ix]*h_Vz[3]+h_g4[iz][iy][ix]*h_Vz[4]+h_g5[iz][iy][ix]*h_Vz[5]+h_g6[iz][iy][ix]*h_Vz[6]+h_g7[iz][iy][ix]*h_Vz[7]+h_g8[iz][iy][ix]*h_Vz[8]+h_g9[iz][iy][ix]*h_Vz[9]+h_g10[iz][iy][ix]*h_Vz[10]+h_g11[iz][iy][ix]*h_Vz[11]+h_g12[iz][iy][ix]*h_Vz[12]+h_g13[iz][iy][ix]*h_Vz[13]+h_g14[iz][iy][ix]*h_Vz[14]+h_g15[iz][iy][ix]*h_Vz[15]+h_g16[iz][iy][ix]*h_Vz[16]+h_g17[iz][iy][ix]*h_Vz[17]+h_g18[iz][iy][ix]*h_Vz[18];
  
  sum1 = h_g0[iz][iy][ix]+h_g1[iz][iy][ix]+h_g2[iz][iy][ix]+h_g3[iz][iy][ix]+h_g4[iz][iy][ix]+h_g5[iz][iy][ix]+h_g6[iz][iy][ix]+h_g7[iz][iy][ix]+h_g8[iz][iy][ix]+h_g9[iz][iy][ix]+h_g10[iz][iy][ix]+h_g11[iz][iy][ix]+h_g12[iz][iy][ix]+h_g13[iz][iy][ix]+h_g14[iz][iy][ix]+h_g15[iz][iy][ix]+h_g16[iz][iy][ix]+h_g17[iz][iy][ix]+h_g18[iz][iy][ix];
  
  return 3.*sum2/(3.*sum1 + 3.*h_P(ix,iy,iz));
}
float LatticeBoltzmann::h_gamma(float Ux0,float Uy0,float Uz0){
  float U2;
  U2 = Ux0*Ux0 + Uy0*Uy0 + Uz0*Uz0;
  return 1./sqrt(1.-(U2/(C*C)));
}
float LatticeBoltzmann::h_n(int ix,int iy,int iz,float Ux0,float Uy0,float Uz0){
  float sum = 0;
  sum = h_f0[iz][iy][ix]+h_f1[iz][iy][ix]+h_f2[iz][iy][ix]+h_f3[iz][iy][ix]+h_f4[iz][iy][ix]+h_f5[iz][iy][ix]+h_f6[iz][iy][ix]+h_f7[iz][iy][ix]+h_f8[iz][iy][ix]+h_f9[iz][iy][ix]+h_f10[iz][iy][ix]+h_f11[iz][iy][ix]+h_f12[iz][iy][ix]+h_f13[iz][iy][ix]+h_f14[iz][iy][ix]+h_f15[iz][iy][ix]+h_f16[iz][iy][ix]+h_f17[iz][iy][ix]+h_f18[iz][iy][ix];
  return sum/h_gamma(Ux0,Uy0,Uz0);
}
float LatticeBoltzmann::h_P(int ix,int iy,int iz){
  int i,j; float sum1=0, sum2=0;
  float g_aux[19] = {h_g0[iz][iy][ix],h_g1[iz][iy][ix],h_g2[iz][iy][ix],h_g3[iz][iy][ix],h_g4[iz][iy][ix],h_g5[iz][iy][ix],h_g6[iz][iy][ix],h_g7[iz][iy][ix],h_g8[iz][iy][ix],h_g9[iz][iy][ix],h_g10[iz][iy][ix],h_g11[iz][iy][ix],h_g12[iz][iy][ix],h_g13[iz][iy][ix],h_g14[iz][iy][ix],h_g15[iz][iy][ix],h_g16[iz][iy][ix],h_g17[iz][iy][ix],h_g18[iz][iy][ix]};

  for(i=0;i<Q;i++){
    sum1 += g_aux[i];
    for(j=0;j<Q;j++){
      sum2 += (g_aux[i]*g_aux[j]*(h_Vx[i]*h_Vx[j]+h_Vy[i]*h_Vy[j]+h_Vz[i]*h_Vz[j]));
    }
  }
  return -(1./3.)*sum1 + (1./3.)*sqrt(-3.*sum2 + 4.*sum1*sum1);
}
float LatticeBoltzmann::h_rho(int ix,int iy,int iz){
  return 3.*h_P(ix,iy,iz);
}
float LatticeBoltzmann::h_feq(int i,float n0,float Ux0,float Uy0,float Uz0){
  float y,U2,UdotV;

  y = h_gamma(Ux0,Uy0,Uz0);
  UdotV = Ux0*h_Vx[i]+Uy0*h_Vy[i]+Uz0*h_Vz[i];
  U2 = Ux0*Ux0 + Uy0*Uy0 + Uz0*Uz0;

  return h_w[i]*n0*y*(1.+3.*UdotV/(cl*cl) + (9./2.)*(UdotV*UdotV)/(cl*cl*cl*cl) - (3./2.)*(U2/(cl*cl)));
}
float LatticeBoltzmann::h_geq(int i,float rho0,float P0,float Ux0,float Uy0,float Uz0){
  float y2,UdotV,U2;

  y2 = h_gamma(Ux0,Uy0,Uz0)*h_gamma(Ux0,Uy0,Uz0);
  UdotV = Ux0*h_Vx[i]+Uy0*h_Vy[i]+Uz0*h_Vz[i];
  U2 = Ux0*Ux0 + Uy0*Uy0 + Uz0*Uz0;
  
  if(i == 0){
    return 3.*P0*h_w[0]*y2*(4. - (2.+ cl*cl)/(y2*cl*cl) - 2.*(U2/(cl*cl)));
  }else{
    return 3.*h_w[i]*P0*y2*( 1./(y2*cl*cl) + 4.*UdotV/(cl*cl) + 6.*(UdotV*UdotV)/(cl*cl*cl*cl) - 2.*(U2/(cl*cl)) );
  }
}

void LatticeBoltzmann::Print(const char * NombreArchivo){
  float Ux0,Uy0,Uz0;
  //Imprimir en un archivo
  ofstream MiArchivo(NombreArchivo);
  ofstream X_Y("X_Y_cut.dat");
  ofstream X_Z("X_Z_cut.dat");
  Show();
  for(int ix=0;ix<M;ix++){
    for(int iy=0;iy<N;iy++)
      for(int iz=0;iz<W;iz++){
        Ux0=h_Ux(ix,iy,iz);
        Uy0=h_Uy(ix,iy,iz);
        Uz0=h_Uz(ix,iy,iz);
        MiArchivo<<ix<<" "<< iy << " " << iz << " " << h_n(ix,iy,iz,Ux0,Uy0,Uz0)<<" "<<h_P(ix,iy,iz)/2.495e-7<<endl;
        if(iz == int(W*0.3)){
          X_Y<<ix<<" "<< iy << " " << h_P(ix,iy,iz)/2.495e-7 << endl;
        }else if(iy == int(N*0.3)){
          X_Z<<ix<<" "<< iz << " " << h_P(ix,iy,iz)/2.495e-7 << endl;
        }
	}
    MiArchivo<<endl;
    X_Y<<endl;
    X_Z<<endl;
  }
  MiArchivo.close();
  X_Y.close();
  X_Z.close();
}


//----------------------------------------------------------

int main()
{ LatticeBoltzmann Relativistic_Ang;

  float Ux0 = 0.0;
  float Uy0 = 0.0;
  float Uz0 = 0.0;

  float T = 0.0314;
  float dg = 16;

  float P0 = 2.495e-7;
  float P1 = 1.023e-7;

  float n0 = P0/T;
  float n1 = P1/T;

  float rho0 = 3*n0*T;
  float rho1 = 3*n1*T;

  int t,tmax = 5000;

  Relativistic_Ang.Start(Ux0,Uy0,Uz0,rho0,rho1,n0,n1,P0,P1);
  
  for(t=0;t<tmax;t++){
    Relativistic_Ang.Collision();
    Relativistic_Ang.Advection();
  }
  Relativistic_Ang.Print("data.dat");   
  return 0;
}
